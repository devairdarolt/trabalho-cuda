#include "hip/hip_runtime.h"

#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- VARIÁVEIS GLOBAIS DA PLACA DE VIDEO                                                                                            //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


extern __device__ Data * global_part=NULL; //Array global para guardar os índices de partições préordenadas
extern __device__ long global_nr_part=0;   //Tamanho do array de particoes;
extern __device__  long * global_vet_device=NULL; //Array global para guardar o vetor a ser ordenado
extern __device__  long global_size_vet=0;
extern __device__  long global_nr_nucleos=0;


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES PRIVADAS (Não acessível para o programa main)                                                                          //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int is_sort(long * arr,long n);


__device__ double ceild(double num);

__device__ void sort_subarray(long arr[], long n, long exp) ;

__device__ long get_max_val(long arr[], long n);

__device__ long radix_sort_array(long x);


__device__ void swap(long* a, long* b);

__device__ void heapify(long *arr, long n, long i);

__device__ void heapSort(long *arr, long n);

__device__ long heap_sort_array(long x);


__device__ void intercala (long p, long q, long r, long *v);

__device__ void print_erro(const char *func,const char *msg);

__device__ void print_sucess(const char *func,const char *msg);


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- HEAP SORT                                                                                                                      //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// A utility function to swap two elements
__device__ void swap(long* a, long* b)
{
    long t = *a;
    *a = *b;
    *b = t;
}

// To heapify a subtree rooted with node i which is
// an index in arr[]. n is size of heap
__device__ void heapify(long *arr, long n, long i)
{
    long largest = i; // Initialize largest as root
    long l = 2 * i + 1; // left = 2*i + 1
    long r = 2 * i + 2; // right = 2*i + 2
 
    // If left child is larger than root
    if (l < n && arr[l] > arr[largest])
        largest = l;
 
    // If right child is larger than largest so far
    if (r < n && arr[r] > arr[largest])
        largest = r;
 
    // If largest is not root
    if (largest != i) {
        swap(&arr[i], &arr[largest]);
 
        // Recursively heapify the affected sub-tree
        heapify(arr, n, largest);
    }
}
 
// main function to do heap sort
__device__ void heapSort(long *arr, long n)
{
    // Build heap (rearrange array)
    for (long i = n / 2 - 1; i >= 0; i--)
        heapify(arr, n, i);
 
    // One by one extract an element from heap
    for (long i = n - 1; i > 0; i--) {
        // Move current root to end
        swap(&arr[0], &arr[i]);
 
        // call max heapify on the reduced heap
        heapify(arr, i, 0);
    }
}
 
__device__ long heap_sort_array(long x){
	
	//if(x!=0) return 0; //para facilitar a programação 	
	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	long n =(long) ceild((double)global_size_vet/(double)global_nr_nucleos); // arredonda pra cima
	long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	if((global_size_vet%global_nr_nucleos!=0)&&(x==global_nr_nucleos-1)){	
		n=global_size_vet-a;		
	}
	long b = (a +n)-1;
	global_part[x].a =a;
	global_part[x].b=b;
	global_part[x].n=n;
	

	long *sub_arr =NULL;
	
	sub_arr = &global_vet_device[a];
	heapSort(&sub_arr[0], n);		
	__syncthreads();
	
	//if(x!=1) return 0; //para facilitar a programação 
	is_sort(&sub_arr[0], n);	
	return 1;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- RADIX SORT                                                                                                                     //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ void sort_subarray(long *arr, long n, long exp) 
{ 
	//long output[n]; // output array 
	long *output = (long*)malloc(n * sizeof(long));
	if(output==NULL){
		
		print_erro("sort_subarray","Erro ao alocar memória na placa de vídeo para 'output'");				
	}
	long i, count[10] = { 0 }; 
	
	// Store count of occurrences in count[] 
	for (i = 0; i < n; i++) 
		count[(arr[i] / exp) % 10]++; 

	// Change count[i] so that count[i] now contains actual 
	// position of this digit in output[] 
	for (i = 1; i < 10; i++) 
		count[i] += count[i - 1]; 

	// Build the output array 
	for (i = n - 1; i >= 0; i--) { 
		output[count[(arr[i] / exp) % 10] - 1] = arr[i]; 
		count[(arr[i] / exp) % 10]--; 
	} 

	// Copy the output array to arr[], so that arr[] now 
	// contains sorted numbers according to current digit 
	for (i = 0; i < n; i++){
		arr[i] = output[i]; 
	}
	free(output); 
		
}

__device__ long get_max_val(long *arr, long n) 
{ 
	long mx = arr[0]; 
	for (long i = 1; i < n; i++) 
		if (arr[i] > mx) 
			mx = arr[i]; 
	return mx; 
} 

__device__ long radix_sort_array(long x){
	
	//if(x!=0) return 0; //para facilitar a programação 
	
	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	long n =(long) ceild((double)global_size_vet/(double)global_nr_nucleos); // arredonda pra cima
	long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	if((global_size_vet%global_nr_nucleos!=0)&&(x==global_nr_nucleos-1)){	
		n=global_size_vet-a;		
	}
	long b = (a +n)-1;
	global_part[x].a =a;
	global_part[x].b=b;
	global_part[x].n=n;
	//set do vetor de particoes;	
	//long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	long *sub_arr =NULL;
	/*long *sub_arr = (long *)malloc(n * sizeof(long));// Cria na memória um espaço para um sub_array
	if(!sub_arr){
		printf("\033[0;31m Erro ao alocar sub array\n \e[m");
	}
	*/
	//printf("Part[%d]:n[%d] [%d <= x <= %d]\n",x,n,a,b);
	//sub_array recebe a referencia da posição inicial do vetor global
	sub_arr = &global_vet_device[a];
	
	/*
		long j=0;
		long aux;
		//printf("\n");
		for(long i=a;i<=b;i++,j++){
			aux = global_vet_device[i];
			sub_arr[j]=aux;			
		}
	*/
	long m = get_max_val(&sub_arr[0], n); 
	//iteração para cada dígito, no caso de um long muito grande esse for vai ocorrer 2^32 -> (10 casas) 
	for (long exp = 1; m / exp > 0; exp *= 10) {
		sort_subarray(&sub_arr[0], n, exp); // primeiro faz o sort pelo bit 0, bit 1 ... até bit exp 
	}	

	/*
		j=0;
		for(long i=a;i<=b;i++,j++){
			global_vet_device[i]=sub_arr[j];
		}
	*/

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE KERNEL                                                                                                              //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void GPU_set_globals(long *vet_d, long vet_size,long nthreads){
	//seta as variáveis globais
	global_vet_device = vet_d;
	global_size_vet = vet_size;
	global_nr_nucleos = nthreads;
	global_nr_part = nthreads;
	global_part = (Data *)malloc(nthreads * sizeof(Data));
	if(global_part==NULL){
		print_erro("GPU_set_globals","Erro ao alocar memória para 'global_part' na placa de video");
	}

}

__global__ void GPU_call_sort (long nthreads) {
	long x = threadIdx.x;
	
	
	//Inicia particionamento e ordenação
	if((global_size_vet<2000000)){
		if(x==0){
			printf("\nutilizando [radix sort]\n");
		}
		radix_sort_array(x);	//TODO anteriormente
	}else{
		if(x==0){
			printf("\nUtilizando[heap sort]\n");
		}
		heap_sort_array(x);	

	}
	

}
__global__ void GPU_reset(){
	free(global_vet_device);
	free(global_part);
}

__global__ void GPU_print(){
	printf("GPU_print\n");
	if(global_vet_device==NULL||global_size_vet==0||global_vet_device[global_size_vet-1]==0){		
		print_erro("GPU_print","os dados não foram copiados para a memória da placa de video...");
	}
	
	int max_index = global_size_vet;
	if(max_index>20){
		max_index = 20;
	}
	printf("\n%d primeiros:",max_index);
	for(int i=0;i<max_index;i++){
		printf(" %ld ",global_vet_device[i]);
	}
	printf("\n%d ultimos:",max_index);
	long sum=0;
	for(int i=global_size_vet-max_index;i<global_size_vet;i++){
		printf(" %ld ",global_vet_device[i]);
		sum+=global_vet_device[i];
	}	
	
	if(global_vet_device[global_size_vet-1]==NULL||sum==0){		
		print_erro("GPU_print","os dados não foram copiados para a memória da placa...");
	}
	printf("\n");
	
	if(is_sort(&global_vet_device[0],global_size_vet)){		
		print_sucess("GPU_print","VETOR ORDENADO!");
	}else{
		print_erro("GPU_print","VETOR DESORDENADO!");		
	}	
	printf("\n");
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE AUXILIARES                                                                                                          //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ long *criar_vetor_desordenado(long vet_size){	
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	printf("Alocando na memória do host\n");
	long *vet;
	hipHostMalloc((void **) &vet, vet_size*sizeof(long));	
	if(vet==NULL){
		h_print_erro("criar_vetor_desordenado","Erro ao alocar memória 'hipHostMalloc'");
	}
	printf("memória alocada\n");
	//inicia valores do vetor desordenado
	srand(time(0));
	for(long i=0;i<vet_size;i++){
		vet[i]= rand() % 100000;// (0 <= rand <= vet_size)
	}
	printf("Vetor aleatório gerado alocado\n");
	return vet;
}

__host__ void vet_imprimir(long *v,long vet_size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}
	//printf("primeiro elemento:%d\n",v[0]);
	//printf("ultimo elemento:%d\n",v[vet_size-1]);	
	printf("vet_d: ");
	long max_index = vet_size;
	if(max_index>50){
		max_index = 50;
	}
	
	printf("\n%ld primeiros\t",max_index);
	for(long i=0;i<max_index;i++){
		printf(" %ld, ",v[i]);
	}
	if(vet_size>50){
		printf("\n%ld Ultimas\t",max_index);
		for(long i=vet_size-50;i<vet_size;i++){
			printf(" %ld, ",v[i]);
		}		
	}		
	printf("\n");

	int ordenado = h_is_sort(v,vet_size);	
	if(ordenado){
		h_print_sucess("vet_imprimir","ORDENADO!");
		
	}else{
		h_print_erro("vet_imprimir","DESORDENADO!");		
	}	


}

__host__ double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + (double) t.tv_usec / 1000000;
}



__device__ double ceild(double num){	
	long inum = (long)num;
    if (num == (float)inum) {
        return inum;
    }
    return inum + 1;
}

// A função recebe vetores crescentes v[p..q-1] 
// e v[q..r-1] e rearranja v[p..r-1] em ordem 
// crescente.
//Tem um custo maior de espaço pois cada thread cria um vetor de tamanho n
__device__ void intercala (long p, long q, long r, long *v) 
{
   long *w;                                 //  1
   w =(long *)malloc((r-p) * sizeof(long));  //  2
   if(w==NULL){
		print_erro("intercala","Não foi possivel alocar memoria para w");
   }
   long i = p, j = q;                       //  3
   long k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   free (w);                               // 12
}

__global__ void GPU_merge (long nr_thread){
	long x = threadIdx.x;
	//printf("[GPU_merge]\n");
	//if(x!=0)return;
	// |0..3|4..7|||8..9|
	//    0    1     2       3         4
	//    ______     __________      _____ 
	//x:     0           1             2
	long a1 = global_part[x*2].a;
	long b1 = global_part[x*2].b;
	long n1 = global_part[x*2].n;
	long a2,b2,n2;
	long single_part =0;
	
	Data *xData = (Data*)malloc(sizeof(Data));
	if(xData==NULL){
		print_erro("GPU_merge","Erro ao alocar memória na placa de vídeo");		
	}
	if(b1==global_size_vet-1){
		single_part = 1;		
	}
	if(!single_part){
		a2 = global_part[(x*2)+1].a;
		b2 = global_part[(x*2)+1].b;
		n2 = global_part[(x*2)+1].n;

		xData->a=a1;
		xData->b=b2;
		xData->n=n1+n2;				
		//printf("\033[0;34m x[%ld]-{(a1:%ld,b1:%ld),(a2:%ld,b2:%ld)- merge----{a1:%ld,b2:%ld} - n:%ld}\e[m\n",x,a1,b1,a2,b2,xData->a,xData->b,xData->n);		
		intercala(a1,a2,a2+n2,global_vet_device);
		if(!is_sort(&global_vet_device[a1],xData->n)){
			print_erro("GPU_merge","A sub particao não esta ordenada");
		}
		//global_part[x].a=a1;
		//global_part[x].b=b2;
	}else{
		//global_part[x].a=a1;
		//global_part[x].b=b1;
		xData->a=a1;
		xData->b=b1;
		xData->n=(b1+1)-a1;
		//printf("Part[%ld]-{(%ld,%ld) n:%ld- copiado}\n",x,a1,b1,xData->n);		
		if(!is_sort(&global_vet_device[a1],xData->n)){
			print_erro("GPU_merge","A sub particao não esta ordenada");
		}		
	}
	
	__syncthreads();// Quando todas as threads chegarem aqui escolhe uma thread para alocar o vetor de particoes		

	//parte do código executada apenas pela ultima thread	
	if(x==nr_thread-1){
		//printf("Thread:%d reorganizando vetor de particoes...\n",x);
		free(global_part);
		global_part = (Data *)malloc((x+1)*sizeof(Data));
		if(global_part==NULL){
			print_erro("GPU_merge","Erro ao alocar memoria para 'global_part'");
		}
		global_nr_part = x+1;
	}
	__syncthreads();	
	global_part[x]=*xData;
		
}
__device__ void print_erro(const char *func,const char *msg){
	printf("\033[0;31m [%s]--%s\e[m\n",func,msg);
}
__device__ void print_sucess(const char *func,const char *msg){
	printf("\033[0;32m [%s]--%s\e[m\n",func,msg);
}


__device__ int is_sort(long * arr,long n){	
	long ordenado = 0;	
	for(long i=1;i<n;i++){
		if(arr[i-1]>arr[i]){
			//printf("arr[%ld]>arr[%ld]--[%ld,%ld]\n",i-1,i,arr[i-1],arr[i]);
			ordenado ++;			
		}		
	}
	if(ordenado==0){
		//printf("\nSub particao ordenada!\n");
		return 1;
	}else{
		//printf("\nSub particao desordenada, %ld posicoes fora de ordem!\n",ordenado);
		return 0;
	}	
}
//#####################################################################################################
__global__ void GPU_get_d_part(Data *d_part){
	memcpy(d_part,global_part,global_nr_part*sizeof(Data));
	
}
__global__ void GPU_get_nr_part(long *d_nr_part){
	//int erro_memcpy;	
	*d_nr_part = global_nr_part;
	//printf("d_nr_part %ld\n",*d_nr_part);

}

__global__ void GPU_get_global_vet(long *d_vet){
	//d_vet = global_vet_device;
	/* for(int i=0;i<global_size_vet;i++){
		d_vet[i] =global_vet_device[i];
	} */
	//d_vet = global_vet_device;
	memcpy(d_vet,global_vet_device,global_size_vet*sizeof(long));
	printf("d_vet[0] %d\n",d_vet[0]);	
	//memcpy(d_vet,global_vet_device,global_size_vet * sizeof(long));
}

__host__ void h_print_erro(const char *func,const char *msg){
	printf("\033[0;31m [%s]--%s\e[m\n",func,msg);
}

__host__ void h_print_sucess(const char *func,const char *msg){
	printf("\033[0;32m [%s]--%s\e[m\n",func,msg);
}


__host__ int h_is_sort(long * arr,long n){	
	long ordenado = 0;	
	for(long i=1;i<n;i++){
		if(arr[i-1]>arr[i]){
			//printf("arr[%ld]>arr[%ld]--[%ld,%ld]\n",i-1,i,arr[i-1],arr[i]);
			ordenado ++;			
		}		
	}
	if(ordenado==0){
		//printf("\nSub particao ordenada!\n");
		return 1;
	}else{
		//printf("\nSub particao desordenada, %ld posicoes fora de ordem!\n",ordenado);
		return 0;
	}	
}