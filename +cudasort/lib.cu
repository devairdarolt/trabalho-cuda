#include "hip/hip_runtime.h"
#include "lib.h"
#include "lista.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>


__device__ void device_log(char *msg){
	printf("%s\n", msg);
}


//##########################################################################

__device__ Data * global_part=NULL;
__device__ Data global_last_part;
__device__ long global_nr_part=0;
__device__  long * global_vet_device=NULL;
__device__  long global_size_vet=0;
__device__  long global_nr_nucleos=0;

__device__ Lista particoes;// = list_init(sizeof(Data)); // Inicia uma lista para o tipo Data



// FUNÇÕES PRIVADAS (Não acessível para o programa main)
__device__ double ceild(double num);

__device__ void sort_subarray(long arr[], long n, long exp) ;

__device__ long get_max_val(long arr[], long n);

__device__ long sort_array(long x);

__device__ void intercala (long p, long q, long r, long *v);

// CODE
__device__ void sort_subarray(long arr[], long n, long exp) 
{ 
	//long output[n]; // output array 
	long *output = (long*)malloc(n * sizeof(long));
	long i, count[10] = { 0 }; 
	//printf("sort_subarray[arr[0]]:%d\n", arr[0]);
	// Store count of occurrences in count[] 
	for (i = 0; i < n; i++) 
		count[(arr[i] / exp) % 10]++; 

	// Change count[i] so that count[i] now contains actual 
	// position of this digit in output[] 
	for (i = 1; i < 10; i++) 
		count[i] += count[i - 1]; 

	// Build the output array 
	for (i = n - 1; i >= 0; i--) { 
		output[count[(arr[i] / exp) % 10] - 1] = arr[i]; 
		count[(arr[i] / exp) % 10]--; 
	} 

	// Copy the output array to arr[], so that arr[] now 
	// contains sorted numbers according to current digit 
	for (i = 0; i < n; i++){
		arr[i] = output[i]; 
	} 
		
}

__device__ long get_max_val(long arr[], long n) 
{ 
	long mx = arr[0]; 
	for (long i = 1; i < n; i++) 
		if (arr[i] > mx) 
			mx = arr[i]; 
	return mx; 
} 

__device__ long sort_array(long x){
	
	//if(x!=0) return 0; //para facilitar a programação 
	
	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	long n =(long) ceild((double)global_size_vet/(double)global_nr_nucleos); // arredonda pra cima
	long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	if((global_size_vet%global_nr_nucleos!=0)&&(x==global_nr_nucleos-1)){	
		n=global_size_vet-a;		
	}
	long b = (a +n)-1;
	global_part[x].a =a;
	global_part[x].b=b;
	global_part[x].n=n;
	//set do vetor de particoes;	
	//long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	long *sub_arr = (long *)malloc(n * sizeof(long));// Cria na memória um espaço para um sub_array
	if(!sub_arr){
		printf("\033[0;31m Erro ao alocar sub array\n \e[m");
	}
	//printf("Part[%d]:n[%d] [%d <= x <= %d]\n",x,n,a,b);
	//sub_array recebe a referencia da posição inicial do vetor global
	//sub_arr = &global_vet_device[a];
	long j=0;
	long aux;
	//printf("\n");
	for(long i=a;i<=b;i++,j++){
		aux = global_vet_device[i];
		sub_arr[j]=aux;			
	}

	long m = get_max_val(sub_arr, n); 
	//iteração para cada dígito, no caso de um long muito grande esse for vai ocorrer 2^32 -> (10 casas) 
	for (long exp = 1; m / exp > 0; exp *= 10) {
		sort_subarray(&sub_arr[0], n, exp); 
	}	

	j=0;
	for(long i=a;i<=b;i++,j++){
		global_vet_device[i]=sub_arr[j];
	}

	return 0;
}

__global__ void GPU_set_globals(long *vet_d, long vet_size,long nthreads){
	//seta as variáveis globais
	global_vet_device = vet_d;
	global_size_vet = vet_size;
	global_nr_nucleos = nthreads;
	global_part = (Data *)malloc(nthreads * sizeof(Data));

}

__global__ void GPU_sort (long nthreads) {
	long x = threadIdx.x;
	
	
	//Inicia particionamento e ordenação
	sort_array(x);	

}


__host__ long *criar_vetor_desordenado(long *v,long vet_size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}

	hipHostMalloc((void **) &v, vet_size*sizeof(long));	
	//inicia valores do vetor desordenado
	srand(time(0));
	for(long i=0;i<vet_size;i++){
		v[i]= rand() % 10000;// (0 <= rand <= vet_size)
	}
	return v;
}

__host__ void vet_imprimir(long *v,long vet_size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}
	//printf("primeiro elemento:%d\n",v[0]);
	//printf("ultimo elemento:%d\n",v[vet_size-1]);	
	printf("vet_d: ");
	if(1==1){
		printf("\n20 primeiros\n");
		for(long i=0;i<20;i++){
			printf("%10ld,",v[i]);
		}
		printf("\n20 ultimos\n");
		for(long i=vet_size-20;i<vet_size;i++){
			printf("%10ld,",v[i]);
		}
		
	}
	long value = v[0];
	long ordenado = 1;
	
	for(long i=1;i<vet_size;i++){
		if(v[i]<value){
			ordenado =0;
			break;
		}
		value = v[i];
	}
	if(ordenado){
		printf("\nVETOR ORDENADO!\n");
	}else{
		printf("\nVETOR DESORDENADO!\n");
	}	
	printf("\n");


}

__host__ double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + (double) t.tv_usec / 1000000;
}



__device__ double ceild(double num){	
	long inum = (long)num;
    if (num == (float)inum) {
        return inum;
    }
    return inum + 1;
}

// A função recebe vetores crescentes v[p..q-1] 
// e v[q..r-1] e rearranja v[p..r-1] em ordem 
// crescente.
__device__ void intercala (long p, long q, long r, long *v) 
{
   long *w;                                 //  1
   w =(long *)malloc((r-p) * sizeof(long));  //  2
   long i = p, j = q;                       //  3
   long k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   free (w);                               // 12
}
__global__ void GPU_get_nr_partitions(long *d_nr_part){
	long count;
	for(long i=0; i< global_nr_part;i++){
		//if(global_part[i]!=NULL){
		//}
		count++;
	}
	*d_nr_part = count+1;
	

}

__global__ void GPU_merge (long nr_thread){
	long x = threadIdx.x;
	//printf("[GPU_merge]\n");
	//if(x!=0)return;
	// |0..3|4..7|||8..9|
	//    0    1     2       3         4
	//    ______     __________      _____ 
	//x:     0           1             2
	long a1 = global_part[x*2].a;
	long b1 = global_part[x*2].b;
	long a2,b2;
	long single_part =0;
	
	Data *xData = (Data*)malloc(sizeof(Data));
	if(b1==global_size_vet-1){
		single_part = 1;
		
	}
	if(!single_part){
		a2 = global_part[(x*2)+1].a;
		b2 = global_part[(x*2)+1].b;
		xData->a=a1;
		xData->b=b2;
		xData->n=(xData->b+1)-xData->a;
		
		//printf("Part[%d]-{(%d,%d),(%d,%d)- merge----{%3d,%3d}}\n",x,a1,b1,a2,b2,xData->a,xData->b);		
		intercala(a1,a2,b2+1,global_vet_device);

		//global_part[x].a=a1;
		//global_part[x].b=b2;
	}else{
		//global_part[x].a=a1;
		//global_part[x].b=b1;
		xData->a=a1;
		xData->b=b1;
		//printf("Part[%d]-{(%d,%d)- copiado}\n",x,a1,b1);		
		//free(global_part[x+1]);
	}
	__syncthreads();		
	//parte do código executada apenas pela ultima thread
	
	if(x==nr_thread-1){
		//printf("Thread:%d reorganizando vetor de particoes...\n",x);
		free(global_part);
		global_part = (Data *)malloc((x+1)*sizeof(Data));
		global_nr_part = x+1;
	}
	__syncthreads();	
	global_part[x]=*xData;
		
}

__global__ void GPU_reset(){
	free(global_vet_device);
	free(global_part);
}

__global__ void GPU_print(){
	printf("\n20 primeiros:");
	for(int i=0;i<20;i++){
		printf("%9ld",global_vet_device[i]);
	}
	printf("\n20 ultimos:");
	for(int i=global_size_vet-20;i<global_size_vet;i++){
		printf("%9ld",global_vet_device[i]);
	}
	printf("\n");
}