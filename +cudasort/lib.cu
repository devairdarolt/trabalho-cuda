#include "hip/hip_runtime.h"
#include "lib.h"
#include "lista.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>


__device__ void device_log(char *msg){
	printf("%s\n", msg);
}

typedef struct Data{
	int a;
	int b;
}Data;
//##########################################################################
__device__  int * global_vet_device=NULL;
__device__  int global_size_vet=0;
__device__  int global_nr_nucleos=0;

__device__ Lista particoes;// = list_init(sizeof(Data)); // Inicia uma lista para o tipo Data



// FUNÇÕES PRIVADAS (Não acessível para o programa main)
__device__ double ceild(double num);

__device__ void sort_subarray(int arr[], int n, int exp) ;

__device__ int get_max_val(int arr[], int n);

__device__ int sort_array(int x);

__device__ void intercala (int p, int q, int r, int *v);

// CODE
__device__ void sort_subarray(int arr[], int n, int exp) 
{ 
	//int output[n]; // output array 
	int *output = (int*)malloc(n * sizeof(int));
	int i, count[10] = { 0 }; 
	//printf("sort_subarray[arr[0]]:%d\n", arr[0]);
	// Store count of occurrences in count[] 
	for (i = 0; i < n; i++) 
		count[(arr[i] / exp) % 10]++; 

	// Change count[i] so that count[i] now contains actual 
	// position of this digit in output[] 
	for (i = 1; i < 10; i++) 
		count[i] += count[i - 1]; 

	// Build the output array 
	for (i = n - 1; i >= 0; i--) { 
		output[count[(arr[i] / exp) % 10] - 1] = arr[i]; 
		count[(arr[i] / exp) % 10]--; 
	} 

	// Copy the output array to arr[], so that arr[] now 
	// contains sorted numbers according to current digit 
	for (i = 0; i < n; i++){
		arr[i] = output[i]; 
	} 
		
}

__device__ int get_max_val(int arr[], int n) 
{ 
	int mx = arr[0]; 
	for (int i = 1; i < n; i++) 
		if (arr[i] > mx) 
			mx = arr[i]; 
	return mx; 
} 

__device__ int sort_array(int x){
	
	//if(x!=0) return 0; //para facilitar a programação 
	
	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	int n =(int) ceild((double)global_size_vet/(double)global_nr_nucleos); // arredonda pra cima
	int a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	if((global_size_vet%global_nr_nucleos!=0)&&(x==global_nr_nucleos-1)){	
		n=global_size_vet-a;		
	}
	int b = a +n;

	//int a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	int *sub_arr = (int *)malloc(n * sizeof(int));// Cria na memória um espaço para um sub_array

	printf("Part[%d]:n[%d] [%d <= x < %d]\n",x,n,a,b);
	//sub_array recebe a referencia da posição inicial do vetor global
	//sub_arr = &global_vet_device[a];
	int j=0;
	int aux;
	printf("\n");
	for(int i=a;i<b;i++,j++){
		aux = global_vet_device[i];
		sub_arr[j]=aux;			
	}

	int m = get_max_val(sub_arr, b-a); 
	//iteração para cada dígito, no caso de um int muito grande esse for vai ocorrer 2^32 -> (10 casas) 
	for (int exp = 1; m / exp > 0; exp *= 10) {
		sort_subarray(&sub_arr[0], b-a, exp); 
	}	

	j=0;
	for(int i=a;i<b;i++,j++){
		global_vet_device[i]=sub_arr[j];
	}

	return 0;
}


__global__ void GPU_sort (int *vet_d, int vet_size,int nthreads) {
	int x = threadIdx.x;
	
	//seta as variáveis globais
	global_vet_device = vet_d;
	global_size_vet = vet_size;
	global_nr_nucleos = nthreads;

	
	//Inicia particionamento e ordenação
	sort_array(x);	

}

//############################################################################################
__host__ int *criar_vetor_desordenado(int *v,int vet_size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}

	hipHostMalloc((void **) &v, vet_size*sizeof(int));	
	//inicia valores do vetor desordenado
	for(int i=0;i<vet_size;i++){
		v[i]= rand() % 1000;// (0 <= rand <= vet_size)
	}
	return v;
}

__host__ void vet_imprimir(int *v,int vet_size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}

	printf("primeiro elemento:%d\n",v[0]);
	printf("ultimo elemento:%d\n",v[vet_size-1]);
	printf("Impressão truncada em 100\n");
	/*
	for(int i=0;i<vet_size;i++){
		if(vet_size%(vet_size/10)==0){
			printf("v[%d]:%d\n",i,v[i]);			
		}		
	}*/
		
	
	printf("\n");


}

__host__ double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + (double) t.tv_usec / 1000000;
}



__device__ double ceild(double num){	
	int inum = (int)num;
    if (num == (float)inum) {
        return inum;
    }
    return inum + 1;
}

// A função recebe vetores crescentes v[p..q-1] 
// e v[q..r-1] e rearranja v[p..r-1] em ordem 
// crescente.
__device__ void intercala (int p, int q, int r, int *v) 
{
   int *w;                                 //  1
   w =(int *)malloc((r-p) * sizeof(int));  //  2
   int i = p, j = q;                       //  3
   int k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   free (w);                               // 12
}

__global__ void GPU_merge (int *vet_d, int vet_size,int nthreads){
	int x = threadIdx.x;
	if(x!=0)return;
	printf("[GPU_merge] global_size_vet:%3d\n",global_size_vet);
	printf("[GPU_merge] nthreads:%3d\n",nthreads);
	int m = ceild((double)vet_size/(double)2);//Divide o vetor em duas partes [0][1][2] [3][4]
	int n = vet_size - m;
	int *vet1 = &vet_d[0];
	int *vet2 = &vet_d[m];
	printf("1[%d--%d]\n",0,m-1);
	printf("2[%d--%d]\n",m,vet_size-1);

	intercala(0,m,vet_size,vet_d);

	printf("vet1:\n");
	for(int k=0;k<m;k++){
		printf("%4d",vet_d[k]);
	}
	printf("\n");

	printf("vet2:\n");
	for(int k=m;k<vet_size;k++){
		printf("%4d",vet_d[k]);
	}
	printf("\n");

	printf("vet ordenado:\n");
	for(int k=0;k<vet_size;k++){
		printf("%4d",vet_d[k]);
	}
	printf("\n");
	//merge(vet1,m,vet2,n);
	
}