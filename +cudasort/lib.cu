#include "hip/hip_runtime.h"
#include "lib.h"
__device__  int * global_vet_device=NULL;
__device__  int global_size_vet=0;
__device__  int global_nr_nucleos=0;



__device__ void sort_subarray(int arr[], int n, int exp) 
{ 

	//int output[n]; // output array 
	int *output = (int*)malloc(n * sizeof(int));
	int i, count[10] = { 0 }; 
	//printf("sort_subarray[arr[0]]:%d\n", arr[0]);
	// Store count of occurrences in count[] 
	for (i = 0; i < n; i++) 
		count[(arr[i] / exp) % 10]++; 

	// Change count[i] so that count[i] now contains actual 
	// position of this digit in output[] 
	for (i = 1; i < 10; i++) 
		count[i] += count[i - 1]; 

	// Build the output array 
	for (i = n - 1; i >= 0; i--) { 
		output[count[(arr[i] / exp) % 10] - 1] = arr[i]; 
		count[(arr[i] / exp) % 10]--; 
	} 

	// Copy the output array to arr[], so that arr[] now 
	// contains sorted numbers according to current digit 
	for (i = 0; i < n; i++){
		arr[i] = output[i]; 
	} 
		
}

__device__ int get_max_val(int arr[], int n) 
{ 
	int mx = arr[0]; 
	for (int i = 1; i < n; i++) 
		if (arr[i] > mx) 
			mx = arr[i]; 
	return mx; 
} 

__device__ int sort_array(int x){
	
	//if(x!=0) return 0; //para facilitar a programação 

		
	//printf("CUDA core [%d]\n",x);

	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	int n = global_size_vet/global_nr_nucleos; // n = sub_arr_size	
	int *sub_arr = (int *)malloc(n * sizeof(int));// Cria na memória um espaço para um sub_array
	int a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	
	//sub_array recebe a referencia da posição inicial do vetor global
	sub_arr = &global_vet_device[a];
	//memcpy(&sub_arr[0],&global_vet_device[a],sizeof(int)*n);
	
	int m = get_max_val(&sub_arr[0], n); 
	//iteração para cada dígito, no caso de um int muito grande esse for vai ocorrer 2^32 -> (10 casas) 
	for (int exp = 1; m / exp > 0; exp *= 10) {
		sort_subarray(&sub_arr[0], n, exp); 
	}	
	
	return 0;

}


__global__ void GPU_sort (int *vet_d, int vet_size,int nthreads) {
	int x = threadIdx.x;
	
	//seta as variáveis globais
	global_vet_device = vet_d;
	global_size_vet = vet_size;
	global_nr_nucleos = nthreads;

	
	//Inicia particionamento e ordenação
	sort_array(x);	

}

//############################################################################################
__host__ int *criar_vetor_desordenado(int *v,int vet_size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	

	hipHostMalloc((void **) &v, vet_size*sizeof(int));
	
	//inicia valores do vetor desordenado
	for(int i=0;i<vet_size;i++){
		v[i]= rand() % 1000;// (0 <= rand <= vet_size)
	}
	return v;
}

__host__ void vet_imprimir(int *v,int vet_size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}

	printf("primeiro elemento:%d\n",v[0]);
	printf("ultimo elemento:%d\n",v[vet_size-1]);
	printf("Impressão truncada em 100\n");
	/*for(int i=0;i<10;i++){
		if(vet_size%(vet_size/8)==0){
			printf("v[%d]:%d\n",i,v[i]);			
		}		
	}*/	
		
	
	printf("\n");


}

__host__ double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + (double) t.tv_usec / 1000000;
}






































/*__global__ void GPU_sort (int *vet_d, int vet_size,int nthreads) {
   

   int k = threadIdx.x;   
   printf("Nucleo %d\n",k );
   int part = vet_size / nthreads; //== cada trede ordenara quatro posições do vetor[40]
   
   /**
		0 < i=0 < 4 .... 4 < i=1 < 8 .... 8 < i=2 < 12 ... 12 < i=3 < 18
   /
   int a = k*part;
   int b = k*part+part;
   int i=0,j=0;
   int min_idx=0,temp;
   for(i=a;i<b;i++){
   		min_idx = i;
   		for(j=i+1;j<b;j++){
   			if(vet_d[j]<vet_d[min_idx]){
   				min_idx = j;
   			}
   		}
   		temp = 0;
   		temp = vet_d[min_idx];
   		vet_d[min_idx] = vet_d[i];
   		vet_d[i] = temp;	
   }

   
   
}*/