#include "hip/hip_runtime.h"
#include "lib.h"
#include "lista.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>


__device__ void device_log(char *msg){
	printf("%s\n", msg);
}


//##########################################################################
__device__ Data * global_part=NULL;
__device__ Data global_last_part;
__device__ int global_nr_part=0;
__device__  int * global_vet_device=NULL;
__device__  int global_size_vet=0;
__device__  int global_nr_nucleos=0;

__device__ Lista particoes;// = list_init(sizeof(Data)); // Inicia uma lista para o tipo Data



// FUNÇÕES PRIVADAS (Não acessível para o programa main)
__device__ double ceild(double num);

__device__ void sort_subarray(int arr[], int n, int exp) ;

__device__ int get_max_val(int arr[], int n);

__device__ int sort_array(int x);

__device__ void intercala (int p, int q, int r, int *v);

// CODE
__device__ void sort_subarray(int arr[], int n, int exp) 
{ 
	//int output[n]; // output array 
	int *output = (int*)malloc(n * sizeof(int));
	int i, count[10] = { 0 }; 
	//printf("sort_subarray[arr[0]]:%d\n", arr[0]);
	// Store count of occurrences in count[] 
	for (i = 0; i < n; i++) 
		count[(arr[i] / exp) % 10]++; 

	// Change count[i] so that count[i] now contains actual 
	// position of this digit in output[] 
	for (i = 1; i < 10; i++) 
		count[i] += count[i - 1]; 

	// Build the output array 
	for (i = n - 1; i >= 0; i--) { 
		output[count[(arr[i] / exp) % 10] - 1] = arr[i]; 
		count[(arr[i] / exp) % 10]--; 
	} 

	// Copy the output array to arr[], so that arr[] now 
	// contains sorted numbers according to current digit 
	for (i = 0; i < n; i++){
		arr[i] = output[i]; 
	} 
		
}

__device__ int get_max_val(int arr[], int n) 
{ 
	int mx = arr[0]; 
	for (int i = 1; i < n; i++) 
		if (arr[i] > mx) 
			mx = arr[i]; 
	return mx; 
} 

__device__ int sort_array(int x){
	
	//if(x!=0) return 0; //para facilitar a programação 
	
	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	int n =(int) ceild((double)global_size_vet/(double)global_nr_nucleos); // arredonda pra cima
	int a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	if((global_size_vet%global_nr_nucleos!=0)&&(x==global_nr_nucleos-1)){	
		n=global_size_vet-a;		
	}
	int b = (a +n)-1;
	global_part[x].a =a;
	global_part[x].b=b;
	global_part[x].n=n;
	//set do vetor de particoes;	
	//int a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	int *sub_arr = (int *)malloc(n * sizeof(int));// Cria na memória um espaço para um sub_array

	printf("Part[%d]:n[%d] [%d <= x <= %d]\n",x,n,a,b);
	//sub_array recebe a referencia da posição inicial do vetor global
	//sub_arr = &global_vet_device[a];
	int j=0;
	int aux;
	//printf("\n");
	for(int i=a;i<=b;i++,j++){
		aux = global_vet_device[i];
		sub_arr[j]=aux;			
	}

	int m = get_max_val(sub_arr, n); 
	//iteração para cada dígito, no caso de um int muito grande esse for vai ocorrer 2^32 -> (10 casas) 
	for (int exp = 1; m / exp > 0; exp *= 10) {
		sort_subarray(&sub_arr[0], n, exp); 
	}	

	j=0;
	for(int i=a;i<=b;i++,j++){
		global_vet_device[i]=sub_arr[j];
	}

	return 0;
}

__global__ void GPU_set_globals(int *vet_d, int vet_size,int nthreads){
	//seta as variáveis globais
	global_vet_device = vet_d;
	global_size_vet = vet_size;
	global_nr_nucleos = nthreads;
	global_part = (Data *)malloc(nthreads * sizeof(Data));

}

__global__ void GPU_sort (int nthreads) {
	int x = threadIdx.x;
	
	
	//Inicia particionamento e ordenação
	sort_array(x);	

}


__host__ int *criar_vetor_desordenado(int *v,int vet_size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}

	hipHostMalloc((void **) &v, vet_size*sizeof(int));	
	//inicia valores do vetor desordenado
	srand(time(0));
	for(int i=0;i<vet_size;i++){
		v[i]= rand() % 10000;// (0 <= rand <= vet_size)
	}
	return v;
}

__host__ void vet_imprimir(int *v,int vet_size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}
	//printf("primeiro elemento:%d\n",v[0]);
	//printf("ultimo elemento:%d\n",v[vet_size-1]);	
	printf("vet_d: ");
	if(1==1){
		if(vet_size<=500){

			for(int i=0;i<vet_size;i++){
				printf("%10d,",v[i]);
			}
		}
	}
	int value = v[0];
	int ordenado = 1;
	
	for(int i=1;i<vet_size;i++){
		if(v[i]<value){
			ordenado =0;
			break;
		}
		value = v[i];
	}
	if(ordenado){
		printf("\nVETOR ORDENADO!\n");
	}else{
		printf("\nVETOR DESORDENADO!\n");
	}	
	printf("\n");


}

__host__ double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + (double) t.tv_usec / 1000000;
}



__device__ double ceild(double num){	
	int inum = (int)num;
    if (num == (float)inum) {
        return inum;
    }
    return inum + 1;
}

// A função recebe vetores crescentes v[p..q-1] 
// e v[q..r-1] e rearranja v[p..r-1] em ordem 
// crescente.
__device__ void intercala (int p, int q, int r, int *v) 
{
   int *w;                                 //  1
   w =(int *)malloc((r-p) * sizeof(int));  //  2
   int i = p, j = q;                       //  3
   int k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   free (w);                               // 12
}
__global__ void GPU_get_nr_partitions(int *d_nr_part){
	int count;
	for(int i=0; i< global_nr_part;i++){
		//if(global_part[i]!=NULL){
		//}
		count++;
	}
	*d_nr_part = count+1;
	

}

__global__ void GPU_merge (int nr_thread){
	int x = threadIdx.x;
	//printf("[GPU_merge]\n");
	//if(x!=0)return;
	// |0..3|4..7|||8..9|
	//    0    1     2       3         4
	//    ______     __________      _____ 
	//x:     0           1             2
	int a1 = global_part[x*2].a;
	int b1 = global_part[x*2].b;
	int a2,b2;
	int single_part =0;
	
	Data *xData = (Data*)malloc(sizeof(Data));
	if(b1==global_size_vet-1){
		single_part = 1;
		
	}
	if(!single_part){
		a2 = global_part[(x*2)+1].a;
		b2 = global_part[(x*2)+1].b;
		xData->a=a1;
		xData->b=b2;
		xData->n=(xData->b+1)-xData->a;
		
		printf("Part[%d]-{(%d,%d),(%d,%d)- merge----{%3d,%3d}}\n",x,a1,b1,a2,b2,xData->a,xData->b);		
		intercala(a1,a2,b2+1,global_vet_device);

		//global_part[x].a=a1;
		//global_part[x].b=b2;
	}else{
		//global_part[x].a=a1;
		//global_part[x].b=b1;
		xData->a=a1;
		xData->b=b1;
		printf("Part[%d]-{(%d,%d)- copiado}\n",x,a1,b1);		
		//free(global_part[x+1]);
	}
	__syncthreads();		
	//parte do código executada apenas pela ultima thread
	
	if(x==nr_thread-1){
		//printf("Thread:%d reorganizando vetor de particoes...\n",x);
		free(global_part);
		global_part = (Data *)malloc((x+1)*sizeof(Data));
		global_nr_part = x+1;
	}
	__syncthreads();	
	global_part[x]=*xData;
	/*if(x==0){
		printf("\n Particoes restante:%d\t",global_nr_part);
		for(int i=0;i<global_nr_part;i++){
			printf("(%3d,%3d)",global_part[i].a,global_part[i].b);
		}
		printf("\n");
	}*/

	//printf("[GPU_merge] x:%d[%d--%d)[%d--%d]\n",x,global_part[x*2].a,global_part[x*2].b,global_part[x*2+1].a,global_part[x*2+1].b);
	//printf("[GPU_merge] x:%d[%d--%d)[%d--%d]\n",x,a1,b1,a2,b2);
	//intercala(a1,a2,b2,global_vet_device);
	//change_global_partition(x,global_part[x*2].a,global_part[x*2+1].b);

	
}