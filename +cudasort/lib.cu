#include "hip/hip_runtime.h"

#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>



////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- VARIÁVEIS GLOBAIS DA PLACA DE VIDEO                                                                                            //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


extern __device__ Data * _device_global_partitions=NULL; //Array global para guardar os índices de partições préordenadas
extern __device__ long _device_global_nr_partitions=0;   //Tamanho do array de particoes;
extern __device__  long * _device_global_array=NULL; //Array global para guardar o vetor a ser ordenado
extern __device__  long _device_global_array_size=0;
extern __device__  long _device_global_nr_thread=0;


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES PRIVADAS (Não acessível para o programa main)                                                                          //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int device_check_sorted(long * arr,long n);


__device__ double device_ceild(double num);

__device__ void device_radix_sort_subarray(long arr[], long n, long exp) ;

__device__ long device_radix_get_max_val(long arr[], long n);

__device__ long device_radix_sort_array(long x);


__device__ void device_swap(long* a, long* b);

__device__ void device_heapify(long *arr, long n, long i);

__device__ void device_heap_sort(long *arr, long n);

__device__ long device_heap_sort_array(long x);


__device__ void device_intercala (long p, long q, long r, long *v);

__device__ void device_print_erro(const char *func,const char *msg);

__device__ void device_print_sucess(const char *func,const char *msg);

__device__ int device_is_sort();

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- BUBLLE SORT                                                                                                                    //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__device__ void device_bubble_sort_array(long index) 
{ 			
	if(index<_device_global_array_size){

		long index_a = index;
		long index_b = index_a+1;
		// testa index out of bound array				
		if(index_a < _device_global_array_size && index_b < _device_global_array_size){	
			long *a = &_device_global_array[index_a];			// posicao i
			long *b = &_device_global_array[index_b];			// posicao i+1
			if(*b < *a){				
				device_swap(a,b);		
			}								
		}
	}
	
    
} 

__device__ void device_bubble_sort(long tId) {

	for(int k=0; k<device_ceild(((double)(_device_global_array_size)/((double)2)));k++){
		
		long x=tId,y=0;
		int shift = 0;
		long posicao=0;
		
		for(int i=0;i<device_ceild(((double)_device_global_array_size)/(double)(2*_device_global_nr_thread)); i++,y+=_device_global_nr_thread){
			posicao = (2 * x) + (2 * y) + shift; // y = deslocamento em relação ao y anterior, deslocamento de n threads			
			device_bubble_sort_array(posicao);					
			
		}
		__syncthreads();
		shift = 1; // desloca uma unidade para pegar os ímpares
		y=0;
		
		for(int i=0;i<device_ceild(((double)_device_global_array_size)/(double)(2*_device_global_nr_thread)); i++,y+=_device_global_nr_thread){
			posicao = (2 * x) + (2 * y) + shift; // y = deslocamento em relação ao y anterior, deslocamento de n threads			
			device_bubble_sort_array(posicao);					
			
		}		
		__syncthreads();
	}
	
	
	

}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- HEAP SORT                                                                                                                      //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// A utility function to device_swap two elements
__device__ void device_swap(long* a, long* b)
{
    long t = *a;
    *a = *b;
    *b = t;
}

// To device_heapify a subtree rooted with node i which is
// an index in arr[]. n is size of heap
__device__ void device_heapify(long *arr, long n, long i)
{
    long largest = i; // Initialize largest as root
    long l = 2 * i + 1; // left = 2*i + 1
    long r = 2 * i + 2; // right = 2*i + 2
 
    // If left child is larger than root
    if (l < n && arr[l] > arr[largest])
        largest = l;
 
    // If right child is larger than largest so far
    if (r < n && arr[r] > arr[largest])
        largest = r;
 
    // If largest is not root
    if (largest != i) {
        device_swap(&arr[i], &arr[largest]);
 
        // Recursively device_heapify the affected sub-tree
        device_heapify(arr, n, largest);
    }
}
 
// main function to do heap sort
__device__ void device_heap_sort(long *arr, long n)
{
    // Build heap (rearrange array)
    for (long i = n / 2 - 1; i >= 0; i--)
        device_heapify(arr, n, i);
 
    // One by one extract an element from heap
    for (long i = n - 1; i > 0; i--) {
        // Move current root to end
        device_swap(&arr[0], &arr[i]);
 
        // call max device_heapify on the reduced heap
        device_heapify(arr, i, 0);
    }
}
 
__device__ long device_heap_sort_array(long x){
	
	//if(x!=0) return 0; //para facilitar a programação 	
	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	long n =(long) device_ceild((double)_device_global_array_size/(double)_device_global_nr_thread); // arredonda pra cima
	long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	if((_device_global_array_size%_device_global_nr_thread!=0)&&(x==_device_global_nr_thread-1)){	
		n=_device_global_array_size-a;		
	}
	long b = (a +n)-1;
	_device_global_partitions[x].a =a;
	_device_global_partitions[x].b=b;
	_device_global_partitions[x].n=n;
	

	long *sub_arr =NULL;
	
	sub_arr = &_device_global_array[a];
	device_heap_sort(&sub_arr[0], n);		
	__syncthreads();
	
	//if(x!=1) return 0; //para facilitar a programação 
	device_check_sorted(&sub_arr[0], n);	
	return 1;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- RADIX SORT                                                                                                                     //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ void device_radix_sort_subarray(long *arr, long n, long exp) 
{ 
	//long output[n]; // output array 
	long *output = (long*)malloc(n * sizeof(long));
	if(output==NULL){
		
		device_print_erro("device_radix_sort_subarray","Erro ao alocar memória na placa de vídeo para 'output'");				
	}
	long i, count[10] = { 0 }; 
	
	// Store count of occurrences in count[] 
	for (i = 0; i < n; i++) 
		count[(arr[i] / exp) % 10]++; 

	// Change count[i] so that count[i] now contains actual 
	// position of this digit in output[] 
	for (i = 1; i < 10; i++) 
		count[i] += count[i - 1]; 

	// Build the output array 
	for (i = n - 1; i >= 0; i--) { 
		output[count[(arr[i] / exp) % 10] - 1] = arr[i]; 
		count[(arr[i] / exp) % 10]--; 
	} 

	// Copy the output array to arr[], so that arr[] now 
	// contains sorted numbers according to current digit 
	for (i = 0; i < n; i++){
		arr[i] = output[i]; 
	}
	free(output); 
		
}

__device__ long device_radix_get_max_val(long *arr, long n) 
{ 
	long mx = arr[0]; 
	for (long i = 1; i < n; i++) 
		if (arr[i] > mx) 
			mx = arr[i]; 
	return mx; 
} 

__device__ long device_radix_sort_array(long x){
	
	//if(x!=0) return 0; //para facilitar a programação 
	
	// 0 <= x=0 < 5 ... 5 <= x=1 <= 10
	long n =(long) device_ceild((double)_device_global_array_size/(double)_device_global_nr_thread); // arredonda pra cima
	long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	if((_device_global_array_size%_device_global_nr_thread!=0)&&(x==_device_global_nr_thread-1)){	
		n=_device_global_array_size-a;		
	}
	long b = (a +n)-1;
	_device_global_partitions[x].a =a;
	_device_global_partitions[x].b=b;
	_device_global_partitions[x].n=n;
	//set do vetor de particoes;	
	//long a = x * n; // if x=0 -> a=0 ... if x=1 --> a=5...  if x=10 --> a = 50
	long *sub_arr =NULL;
	/*long *sub_arr = (long *)malloc(n * sizeof(long));// Cria na memória um espaço para um sub_array
	if(!sub_arr){
		printf("\033[0;31m Erro ao alocar sub array\n \e[m");
	}
	*/
	//printf("Part[%d]:n[%d] [%d <= x <= %d]\n",x,n,a,b);
	//sub_array recebe a referencia da posição inicial do vetor global
	sub_arr = &_device_global_array[a];
	
	/*
		long j=0;
		long aux;
		//printf("\n");
		for(long i=a;i<=b;i++,j++){
			aux = _device_global_array[i];
			sub_arr[j]=aux;			
		}
	*/
	long m = device_radix_get_max_val(&sub_arr[0], n); 
	//iteração para cada dígito, no caso de um long muito grande esse for vai ocorrer 2^32 -> (10 casas) 
	for (long exp = 1; m / exp > 0; exp *= 10) {
		device_radix_sort_subarray(&sub_arr[0], n, exp); // primeiro faz o sort pelo bit 0, bit 1 ... até bit exp 
	}	

	/*
		j=0;
		for(long i=a;i<=b;i++,j++){
			_device_global_array[i]=sub_arr[j];
		}
	*/

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE KERNEL                                                                                                              //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void KERNEL_set_globals(long *vet_d, long vet_size,long nthreads){
	//seta as variáveis globais
	_device_global_array = vet_d;
	_device_global_array_size = vet_size;
	_device_global_nr_thread = nthreads;
	_device_global_nr_partitions = nthreads;
	_device_global_partitions = (Data *)malloc(nthreads * sizeof(Data));
	if(_device_global_partitions==NULL){
		device_print_erro("KERNEL_set_globals","Erro ao alocar memória para '_device_global_partitions' na placa de video");
	}

}

__global__ void KERNEL_call_sort (long nthreads,int opc) {
	long tId = threadIdx.x;
	
	switch(opc){

		case CUDA_BUBBLE:
			if(tId==0){
				printf("\nUtilizando[bubble sort]\n");
			}
			device_bubble_sort(tId);
			break;

		case CUDA_HEAP:
			if(tId==0){
				printf("\nUtilizando[heap sort]\n");
			}
			device_heap_sort_array(tId);
			break;


	}
	

	
	
	
	
	/* for(long k=0;k< nthreads*_device_global_array_size ;k++){
		for(long shift=0;shift<2;shift++){
			//enquanto não estiver 
			int iteracoes_per_array = device_ceild((double)_device_global_array_size/2*nthreads); ///  10 / (2*2) = 10/4 = 3.
			//printf("iteracoes_per_array [%d]\n",iteracoes_per_array);									     
			for(int i=0;i<iteracoes_per_array;i++){			//												 x0       x1      x0        x1      x0       x1 === iterações para fazer o array todo(vezes)
				device_bubble_sort_array(x,shift);			//executa sobre todo o array fazendo swap entre [0][1] - [2][3] - [4][5] - [6][7] - [8][9]   NULL
				if(device_is_sort())return;
				x+=nthreads;				
			}
			x = threadIdx.x;
		}
	} */
	
	//Inicia particionamento e ordenação
	/* if((_device_global_array_size<1000000)){
		if(x==0){
			printf("\nutilizando [radix sort]\n");
		}
		device_radix_sort_array(x);	//TODO anteriormente
	}else{
		if(x==0){
			printf("\nUtilizando[heap sort]\n");
		}
		device_heap_sort_array(x);	

	} */
	

}
__global__ void KERNEL_reset(){
	free(_device_global_array);
	free(_device_global_partitions);
}

__global__ void KERNEL_print_array(){
	printf("KERNEL_print_array\n");
	if(_device_global_array==NULL||_device_global_array_size==0||_device_global_array[_device_global_array_size-1]==0){		
		device_print_erro("KERNEL_print_array","os dados não foram copiados para a memória da placa de video...");
	}
	
	int max_index = _device_global_array_size;
	if(max_index>10){
		max_index = 10;
	}
	printf("\n%d primeiros:",max_index);
	for(int i=0;i<max_index;i++){
		printf(" %ld ",_device_global_array[i]);
	}
	printf("\n%d ultimos:",max_index);
	long sum=0;
	for(int i=_device_global_array_size-max_index;i<_device_global_array_size;i++){
		printf(" %ld ",_device_global_array[i]);
		sum+=_device_global_array[i];
	}	
	
	if(_device_global_array[_device_global_array_size-1]==NULL||sum==0){		
		device_print_erro("KERNEL_print_array","os dados não foram copiados para a memória da placa...");
	}
	printf("\n");
	/* 
	if(device_check_sorted(&_device_global_array[0],_device_global_array_size)){		
		device_print_sucess("KERNEL_print_array","VETOR ORDENADO!");
	}else{
		device_print_erro("KERNEL_print_array","VETOR DESORDENADO!");		
	}	
	printf("\n"); */
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE AUXILIARES                                                                                                          //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ void vet_imprimir(long *v,long vet_size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}
	//printf("primeiro elemento:%d\n",v[0]);
	//printf("ultimo elemento:%d\n",v[vet_size-1]);	
	
	long max_index = vet_size;
	if(max_index>10){
		max_index = 10;
	}
	
	printf("\n%ld primeiros:   ",max_index);
	for(long i=0;i<max_index;i++){
		printf(" %ld, ",v[i]);
	}
	if(vet_size>50){
		printf("\n%ld Ultimos:  ",max_index);
		for(long i=vet_size-max_index;i<vet_size;i++){
			printf(" %ld, ",v[i]);
		}		
	}		
	printf("\n");
	/* 
	int ordenado = h_is_sort(v,vet_size);	
	if(ordenado){
		host_print_sucess("vet_imprimir","ORDENADO!");
		
	}else{
		host_print_erro("vet_imprimir","DESORDENADO!");		
	} */	


}

__host__ double wtime() {
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec + (double) t.tv_usec / 1000000;
}



__device__ double device_ceild(double num){	
	long inum = (long)num;
    if (num == (float)inum) {
        return inum;
    }
    return inum + 1;
}

// A função recebe vetores crescentes v[p..q-1] 
// e v[q..r-1] e rearranja v[p..r-1] em ordem 
// crescente.
//Tem um custo maior de espaço pois cada thread cria um vetor de tamanho n
__device__ void device_intercala (long p, long q, long r, long *v) 
{
   long *w;                                 //  1
   w =(long *)malloc((r-p) * sizeof(long));  //  2
   if(w==NULL){
		device_print_erro("device_intercala","Não foi possivel alocar memoria para w");
   }
   long i = p, j = q;                       //  3
   long k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   free (w);                               // 12
}

__global__ void KERNEL_merge (long nr_thread){
	long x = threadIdx.x;
	//printf("[KERNEL_merge]\n");
	//if(x!=0)return;
	// |0..3|4..7|||8..9|
	//    0    1     2       3         4
	//    ______     __________      _____ 
	//x:     0           1             2
	long a1 = _device_global_partitions[x*2].a;
	long b1 = _device_global_partitions[x*2].b;
	long n1 = _device_global_partitions[x*2].n;
	long a2,b2,n2;
	long single_part =0;
	
	Data *xData = (Data*)malloc(sizeof(Data));
	if(xData==NULL){
		device_print_erro("KERNEL_merge","Erro ao alocar memória na placa de vídeo");		
	}
	if(b1==_device_global_array_size-1){
		single_part = 1;		
	}
	if(!single_part){
		a2 = _device_global_partitions[(x*2)+1].a;
		b2 = _device_global_partitions[(x*2)+1].b;
		n2 = _device_global_partitions[(x*2)+1].n;

		xData->a=a1;
		xData->b=b2;
		xData->n=n1+n2;				
		//printf("\033[0;34m x[%ld]-{(a1:%ld,b1:%ld),(a2:%ld,b2:%ld)- merge----{a1:%ld,b2:%ld} - n:%ld}\e[m\n",x,a1,b1,a2,b2,xData->a,xData->b,xData->n);		
		device_intercala(a1,a2,a2+n2,_device_global_array);
		if(!device_check_sorted(&_device_global_array[a1],xData->n)){
			device_print_erro("KERNEL_merge","A sub particao não esta ordenada");
		}
		//_device_global_partitions[x].a=a1;
		//_device_global_partitions[x].b=b2;
	}else{
		//_device_global_partitions[x].a=a1;
		//_device_global_partitions[x].b=b1;
		xData->a=a1;
		xData->b=b1;
		xData->n=(b1+1)-a1;
		//printf("Part[%ld]-{(%ld,%ld) n:%ld- copiado}\n",x,a1,b1,xData->n);		
		if(!device_check_sorted(&_device_global_array[a1],xData->n)){
			device_print_erro("KERNEL_merge","A sub particao não esta ordenada");
		}		
	}
	
	__syncthreads();// Quando todas as threads chegarem aqui escolhe uma thread para alocar o vetor de particoes		

	//parte do código executada apenas pela ultima thread	
	if(x==nr_thread-1){
		//printf("Thread:%d reorganizando vetor de particoes...\n",x);
		free(_device_global_partitions);
		_device_global_partitions = (Data *)malloc((x+1)*sizeof(Data));
		if(_device_global_partitions==NULL){
			device_print_erro("KERNEL_merge","Erro ao alocar memoria para '_device_global_partitions'");
		}
		_device_global_nr_partitions = x+1;
	}
	__syncthreads();	
	_device_global_partitions[x]=*xData;
		
}
__device__ void device_print_erro(const char *func,const char *msg){
	printf("\033[0;31m [%s]--%s\e[m\n",func,msg);
}
__device__ void device_print_sucess(const char *func,const char *msg){
	printf("\033[0;32m [%s]--%s\e[m\n",func,msg);
}


__device__ int device_check_sorted(long * arr,long n){	
	long ordenado = 0;	
	for(long i=1;i<n;i++){
		if(arr[i-1]>arr[i]){
			//printf("arr[%ld]>arr[%ld]--[%ld,%ld]\n",i-1,i,arr[i-1],arr[i]);
			ordenado ++;			
		}		
	}
	if(ordenado==0){
		//printf("\nSub particao ordenada!\n");
		return 1;
	}else{
		//printf("\nSub particao desordenada, %ld posicoes fora de ordem!\n",ordenado);
		return 0;
	}	
}
//#####################################################################################################
__global__ void KERNEL_get_array_partitions(Data *d_part){
	memcpy(d_part,_device_global_partitions,_device_global_nr_partitions*sizeof(Data));
	
}
__global__ void KERNEL_get_nr_partitions(long *d_nr_part){
	//int erro_memcpy;	
	*d_nr_part = _device_global_nr_partitions;
	//printf("d_nr_part %ld\n",*d_nr_part);

}

__global__ void KERNEL_get_global_array(long *d_vet){
	
	memcpy(d_vet,_device_global_array,_device_global_array_size*sizeof(long));
	
}

__host__ void host_print_erro(const char *func,const char *msg){
	printf("\033[0;31m [%s]--%s\e[m\n",func,msg);
}

__host__ void host_print_sucess(const char *func,const char *msg){
	printf("\033[0;32m [%s]--%s\e[m\n",func,msg);
}


__host__ int h_is_sort(long * arr,long n){	
	long ordenado = 0;	
	for(long i=1;i<n;i++){
		if(arr[i-1]>arr[i]){
			//printf("arr[%ld]>arr[%ld]--[%ld,%ld]\n",i-1,i,arr[i-1],arr[i]);
			ordenado ++;			
		}		
	}
	if(ordenado==0){
		//printf("\nSub particao ordenada!\n");
		return 1;
	}else{
		//printf("\nSub particao desordenada, %ld posicoes fora de ordem!\n",ordenado);
		return 0;
	}	
}

__device__ int device_is_sort(){	
	long * arr = _device_global_array;
	long n = _device_global_array_size;
	long ordenado = 0;	
	for(long i=1;i<n;i++){
		if(arr[i-1]>arr[i]){
			//printf("arr[%ld]>arr[%ld]--[%ld,%ld]\n",i-1,i,arr[i-1],arr[i]);
			ordenado ++;			
		}		
	}
	if(ordenado==0){
		//printf("\nSub particao ordenada!\n");
		return 1;
	}else{
		//printf("\nSub particao desordenada, %ld posicoes fora de ordem!\n",ordenado);
		return 0;
	}	
}