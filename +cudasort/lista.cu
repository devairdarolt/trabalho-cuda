#include "lista.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>

__device__ Lista list_init(int t){
    Lista l;
    l.sizeof_data=t;
	l.root=NULL;
    return l;
}
__device__ int list_is_empty(Lista l){
	return l.root==NULL;
}
__device__ int list_push_first(Lista *l,void *data){	
    Node *p;
	p=(Node *)malloc(sizeof(Node));
	p->data=malloc(l->sizeof_data);
	memcpy(p->data,data,l->sizeof_data);
	p->next=l->root;
	l->root=p;
	p->privious=NULL;
	if(p->next!=NULL)
		p->next->privious=p;
	return 1;	
}
__device__ int list_pop_first(Lista *l, void *data){
	if(list_is_empty(*l)){
		return ERRO_LISTA_VAZIA;
	}else{
		Node *p=l->root;
		memcpy(data,p->data,l->sizeof_data);
		free(p->data);
		l->root=p->next;
		free(p);
		if(l->root!=NULL)
			l->root->privious=NULL;
		return 1;	
	}
	return 0;
}	
__device__ int list_push_last(Lista *l,void *data){	
    if(list_is_empty(*l)){        
		return list_push_first(l,data);
	}else{
		Node *p=l->root;
		Node *novo=(Node *)malloc(sizeof(Node));
		novo->data=malloc(l->sizeof_data);
		memcpy(novo->data,data,l->sizeof_data);
		while(p->next!=NULL){
			p=p->next;
		}
		p->next=novo;
		novo->privious=p;
		novo->next=NULL;
		
	}
	return 1;
}
	
__device__ int list_pop_last(Lista *l,void * data){
	if(list_is_empty(*l)){
		return ERRO_LISTA_VAZIA;
	}else{
		if(l->root->next==NULL){
			return list_pop_first(l,data);
		}else{
			Node *p=l->root;
			while(p->next!=NULL){
				p=p->next;
			}
			memcpy(data,p->data,l->sizeof_data);
			free(p->data);
			p->privious->next=NULL;
			free(p);
			return 1;
		}
	}
}
__device__ int list_push_position(Lista *l, void * data,int pos){
	if(pos<0){
		return ERRO_POSICAO_INVALIDA;
	}else{
		if(pos==0){
			return list_push_first(l,data);
		}else{
			Node *p=l->root;
			int cont=0;
			if(p!=NULL){
				while(p->next!=NULL && cont<pos){
					cont++;
					p=p->next;
				}
			}
			if(cont==pos){
				Node *novo=(Node *)malloc(sizeof(Node));
				novo->data=malloc(l->sizeof_data);
				memcpy(p->data,data,l->sizeof_data);
				novo->next=p;
				novo->privious=p->privious;
				p->privious=novo;
				novo->privious->next=novo;
				return 1;
			}else if(cont==pos-1)
				return list_push_last(l,data);
			 else{
				return ERRO_POSICAO_INVALIDA;
			}
			
		}
	}
}
__device__ int list_pop_position(Lista *l,void *data,int pos){
	if(pos<0){
		return ERRO_POSICAO_INVALIDA;
	}else{
		if(pos==0){
			return list_pop_first(l,data);
		}else{
			Node *p=l->root;
			int cont=0;
			if(p!=NULL){
				while(p->next!=NULL && cont<pos){
					cont++;
					p=p->next;
				}
			}
			if(cont==pos){
				memcpy(data,p->data,l->sizeof_data);
				p->next->privious=p->privious;
				p->privious=p->next;
				free(p->data);
				return 1;
			}
			 else{
				return ERRO_POSICAO_INVALIDA;
			}
			
		}
	}
}		

__device__ int list_get_position(Lista *l,void *data,int pos){
	if(pos<0){
		return ERRO_POSICAO_INVALIDA;
	}else{
		if(pos==0){
			Node *p=l->root;
		    memcpy(data,p->data,l->sizeof_data);
            return 1;
		}else{
			Node *p=l->root;
			int cont=0;
			if(p!=NULL){
				while(p->next!=NULL && cont<pos){
					cont++;
					p=p->next;
				}
			}
			if(cont==pos){
				memcpy(data,p->data,l->sizeof_data);				
				return 1;
			}
			 else{
				return ERRO_POSICAO_INVALIDA;
			}
			
		}
	}
}		