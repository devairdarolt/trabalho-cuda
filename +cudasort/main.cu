#include "hip/hip_runtime.h"
#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>




int main (int argc, char ** argv) {
	long nthreads = 96;
	//long nblocos = 1;
	long vet_size = 100000000; //762.939453

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=%ld e vet_size=%ld\n",nthreads,vet_size);
	} 
	printf("Ordenando %3ld Kbytes\n",(vet_size*4)/1024);
	//vetores do host	
	long *host_vet=NULL;
	host_vet = criar_vetor_desordenado(host_vet,vet_size);//aloca vetor em host
	
	//printf("Vetor desordenado\n");
	printf("Vetor criado..\n");
	vet_imprimir(host_vet,vet_size); 

	

	long *dev_vet =NULL;
	int erro = hipMalloc((void**)&dev_vet,vet_size * sizeof(long));// aloca vetor na memória global da placa
	if(erro){
		printf("\033[0;31m Erro ao alocar memória da placa de video...\n \e[m");
	}
	printf("Dados copiados para a placa de video %3f MB\n",(double)(vet_size*sizeof(long))/1024/1024);
	hipMemcpy (dev_vet, host_vet, vet_size*sizeof(long), hipMemcpyHostToDevice);
	
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	double s_time = wtime();	
		
	
	GPU_set_globals<<<1,1>>>(dev_vet, vet_size,nthreads);		
	hipDeviceSynchronize();
	
	//printf("Teste de copia vetor grande..n:%ld\n",vet_size);
	//GPU_print<<<1,1>>>();
	//hipDeviceSynchronize();

	
	GPU_call_sort<<<1,nthreads>>>(nthreads);	
	hipDeviceSynchronize();	
	GPU_print<<<1,1>>>();
	hipDeviceSynchronize();

	
	while(nthreads>1){
		
		nthreads = ceil((double)nthreads/(double)2);		
		GPU_merge<<<1,nthreads>>>(nthreads);	
		hipDeviceSynchronize();
		//printf("\n\n\nPos GPU_merge\n");
		//GPU_print<<<1,1>>>();
		//hipDeviceSynchronize();
	}		
	/**/
	double e_time = wtime();
	printf("Time:%f (s)\n", e_time-s_time);
	
	printf("\nOpercacao finalizada\n");
	GPU_print<<<1,1>>>();
	hipDeviceSynchronize();
	
	//free(host_vet);
	GPU_reset<<<1,1>>>();
	return 0;
}

