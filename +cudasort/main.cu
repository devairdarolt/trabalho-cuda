#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>


// função executada na GPU
__global__ void sem_nome (int *vet_d, int size) {
   int i = threadIdx.x;

   vet_d[i] = i*100;
   //printf("Sou nucleo %d\n", i);
}


// função executada no HOST

__host__ int *criar_vetor_desordenado(int *v,int size);

__host__ void vet_imprimir(int *v,int size);

int main (int argc, char ** argv) {
	int nthreads = 4;
	int nblocos = 1;

	int size = 40;
	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, size*sizeof(int), hipHostMallocDefault);
	vet_imprimir(vet_desordenado,size); 

	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, size*sizeof(int), hipMemcpyHostToDevice);
	sem_nome<<<1,10>>>(dev_vet, size);
	hipMemcpy (vet_ordenado, dev_vet, size, hipMemcpyDeviceToHost);
	vet_imprimir(vet_ordenado,size); 


	return 0;
}

__host__ int *criar_vetor_desordenado(int *v,int size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	

	hipHostMalloc((void **) &v, size*sizeof(int), hipHostMallocDefault);
	
	//inicia valores do vetor desordenado
	for(int i=0;i<size;i++){
		v[i]= rand() % size;// (0 <= rand <= size)
	}
	return v;
}
__host__ void vet_imprimir(int *v,int size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}

	for(int i=1;i<size-1;i++){
		if(!i%10){
			printf("\n");
		}
		printf("%d\t",v[i]);		
		
	}
	printf("\n");


}
