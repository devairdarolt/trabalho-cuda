#include "hip/hip_runtime.h"
#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include <omp.h>



extern long h_global_nr_part;   //Tamanho do array de particoes;
//Data * h_global_part=NULL; //Array global para guardar os índices de partições préordenadas
//long h_global_nr_part=0;   //Tamanho do array de particoes;
//long * h_global_vet_device=NULL; //Array global para guardar o vetor a ser ordenado
//long h_global_size_vet=0;
//long h_global_nr_nucleos=0;



int main (int argc, char ** argv) {
	long nthreads = 96;
	//long nblocos = 1;
	long vet_size = 100000000; //762.939453

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=%ld e vet_size=%ld\n",nthreads,vet_size);
	} 
	printf("Ordenando %3ld Kbytes\n",(vet_size*4)/1024);
	//vetores do host	
	long *host_vet=NULL;
	host_vet = criar_vetor_desordenado(host_vet,vet_size);//aloca vetor em host
	
	//printf("Vetor desordenado\n");
	printf("Teste imprimir..\n");
	vet_imprimir(host_vet,vet_size); 

	

	long *dev_vet =NULL;
	int erro = hipMalloc((void**)&dev_vet,vet_size * sizeof(long));// aloca vetor na memória global da placa
	if(erro){
		printf("\033[0;31m Erro ao alocar memória da placa de video...\n \e[m");
	}
	printf("Dados copiados para a placa de video %3f MB\n",(double)(vet_size*sizeof(long))/1024/1024);
	hipMemcpy (dev_vet, host_vet, vet_size*sizeof(long), hipMemcpyHostToDevice);
	
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	double s_time = wtime();	
		
	
	GPU_set_globals<<<1,1>>>(dev_vet, vet_size,nthreads);		
	hipDeviceSynchronize();
	
	//printf("Teste de copia vetor grande..n:%ld\n",vet_size);
	//GPU_print<<<1,1>>>();
	//hipDeviceSynchronize();	
	
	GPU_call_sort<<<1,nthreads>>>(nthreads);	
	hipDeviceSynchronize();	
	GPU_print<<<1,1>>>();
	hipDeviceSynchronize();

	//MODO 1 - realiza o merge usando os cuda cores
	/*************  UTILIZA MUITA MEMÓRIA DA PLACA DE VÍDEO
	while(nthreads>1){
		
		nthreads = ceil((double)nthreads/(double)2);		
		GPU_merge<<<1,nthreads>>>(nthreads);	
		hipDeviceSynchronize();
		//printf("\n\n\nPos GPU_merge\n");
		//GPU_print<<<1,1>>>();
		//hipDeviceSynchronize();
	}		
	/**/

	// MODO 2 - realiza o merge utilizando openMP
	///////////// 
	
	cpyGlobalsFromGpu();

	//printf("h_global_nr_part:%ld\n",h_global_nr_part);
	nthreads =100;
	while(nthreads>1){		
		nthreads = ceil((double)nthreads/(double)2);		
		HOST_merge(nthreads);
		

	}
	/////////////


	double e_time = wtime();
	printf("Time:%f (s)\n", e_time-s_time);
	
	printf("\nOpercacao finalizada\n");
	GPU_print<<<1,1>>>();
	hipDeviceSynchronize();
	
	//free(host_vet);
	GPU_reset<<<1,1>>>();


	//TESTE DO OpenMP
	/****************
	printf("____________________________________________________________________________________\n");
	omp_set_num_threads(100);
	int omp_id;
	int omp_n = omp_get_num_threads();
	printf("omp_n:%d\n",omp_id);
	#pragma omp parallel for
	for(int i=0;i<100;i++){
		omp_id = omp_get_thread_num();
		printf("sou omp:%d\n",omp_id);

	}
	printf("____________________________________________________________________________________\n");
	*/
	return 0;
}
__host__ void HOST_merge (long nr_thread){
	int omp_id; 
	omp_set_num_threads(nr_thread); 	// basicamente simula como o merge da placa de video, para usar a mesma logica
	#pragma omp parallel for
	for(int x=0;x<nr_thread;x++){  	// 
		omp_id = omp_get_thread_num();
		printf("x:%d th:%d\n",x,omp_id);
	}
	printf("\n");
}


__host__ void HOST_merge (long nr_thread,long x){
	
	//printf("[GPU_merge]\n");
	//if(x!=0)return;
	// |0..3|4..7|||8..9|
	//    0    1     2       3         4
	//    ______     __________      _____ 
	//x:     0           1             2
	
	/*long a1 = global_part[x*2].a;
	long b1 = global_part[x*2].b;
	long n1 = global_part[x*2].n;
	long a2,b2,n2;
	long single_part =0;
	
	Data *xData = (Data*)malloc(sizeof(Data));
	if(xData==NULL){
		print_erro("GPU_merge","Erro ao alocar memória na placa de vídeo");		
	}
	if(b1==global_size_vet-1){
		single_part = 1;		
	}
	if(!single_part){
		a2 = global_part[(x*2)+1].a;
		b2 = global_part[(x*2)+1].b;
		n2 = global_part[(x*2)+1].n;

		xData->a=a1;
		xData->b=b2;
		xData->n=n1+n2;				
		printf("\033[0;34m x[%ld]-{(a1:%ld,b1:%ld),(a2:%ld,b2:%ld)- merge----{a1:%ld,b2:%ld} - n:%ld}\e[m\n",x,a1,b1,a2,b2,xData->a,xData->b,xData->n);		
		intercala(a1,a2,a2+n2,global_vet_device);
		if(!is_sort(&global_vet_device[a1],xData->n)){
			print_erro("GPU_merge","A sub particao não esta ordenada");
		}
		//global_part[x].a=a1;
		//global_part[x].b=b2;
	}else{
		//global_part[x].a=a1;
		//global_part[x].b=b1;
		xData->a=a1;
		xData->b=b1;
		xData->n=(b1+1)-a1;
		printf("Part[%ld]-{(%ld,%ld) n:%ld- copiado}\n",x,a1,b1,xData->n);		
		if(!is_sort(&global_vet_device[a1],xData->n)){
			print_erro("GPU_merge","A sub particao não esta ordenada");
		}		
	}
	
	__syncthreads();// Quando todas as threads chegarem aqui escolhe uma thread para alocar o vetor de particoes		

	//parte do código executada apenas pela ultima thread	
	if(x==nr_thread-1){
		//printf("Thread:%d reorganizando vetor de particoes...\n",x);
		free(global_part);
		global_part = (Data *)malloc((x+1)*sizeof(Data));
		if(global_part==NULL){
			print_erro("GPU_merge","Erro ao alocar memoria para 'global_part'");
		}
		global_nr_part = x+1;
	}
	__syncthreads();	
	global_part[x]=*xData;
	*/		
}