#include "hip/hip_runtime.h"

#include "lib.h"

#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include <omp.h>





////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- VARIÁVEIS GLOBAIS DO HOST                                                                                                      //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


long h_global_nr_part;   //Tamanho do array de particoes;
Data *h_global_part; //Array global para guardar os índices de partições préordenadas
long *h_global_array; //Array global para guardar o vetor a ser ordenado
long  h_global_array_size;
long h_global_nr_threads;

long *d_global_array; //Array para ser alocado em device
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DO HOST                                                                                                                //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ void host_intercala (long p, long q, long r, long *v);

__host__ void host_print_erro(const char *func,const char *msg);

__host__ void host_print_sucess(const char *func,const char *msg);

__host__ void host_get_global_array();

__host__ void host_get_global_nr_partitions();

__host__ void host_get_global_partitions();

__host__ void cpu_merge();

__host__ void swap(long *xp, long *yp);

__host__ void sequencial_bubble_sort(long *arr, int long);

__host__ double omp_bubble_sort(long *arr,  long n);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE ARQUIVO                                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ int host_make_input_file(char *nome);

__host__ int host_load_input_file(char *nome);





int main (int argc, char ** argv) {
	//INICIALIZA VARIAVEIS GLOBIAS
	h_global_nr_part=0;   //Tamanho do array de particoes;
	h_global_part=NULL; //Array global para guardar os índices de partições préordenadas
	h_global_array=NULL; //Array global para guardar o vetor a ser ordenado
	
	d_global_array=NULL;
	
	h_global_array_size=100000;		
	h_global_nr_threads = 100;
	//long nblocos = 1;
	char nome[] = "caos.map";
	

	
	if (argc == 3 ||argc == 4) {
		h_global_nr_threads = atoi(argv[1]);
		strcpy(nome,argv[2]);
		if(argc == 4){
			h_global_array_size = atoi(argv[3]);	
			host_make_input_file(nome);
		}
		//nome = argv[2];
		//h_global_array_size = atoi(argv[2]);
	}else{
		printf ("./main <h_global_nr_threads> <fileName> --- faz a leitura do arquivo de entrada\n");
		printf ("./main <h_global_nr_threads> <fileName> <size>--- cria um arquivo de leitura com tamanho size.\n");
		return 0;
	} 
	
	if(!((double)h_global_array_size/h_global_nr_threads >= 2)){
		host_print_erro("A quantidade de números precisa ser duas vezes maior que a quantidade de threads","");
		return 0;
	}

	
	host_load_input_file(nome);
	printf("Ordenando vetor de %ld elementos long - %f Kbytes\n",h_global_array_size,((double)h_global_array_size*sizeof(long))/(double)1024);	
	vet_imprimir(h_global_array,h_global_array_size); 	
	printf("\n\n\n");
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////                     SEQUENCIAL BUBBLE SORT                             ////////////////////////////
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	if(h_global_array_size<=1000000){
		double s_time_seq = wtime();
		sequencial_bubble_sort(h_global_array,h_global_array_size);
		double e_time_seq = wtime();
	
		if(h_is_sort(h_global_array,h_global_array_size)){
			vet_imprimir(h_global_array,h_global_array_size);			
			printf("Tempo SEQUENCIAL[%f]\n",e_time_seq-s_time_seq);
			host_print_sucess("SEQUENCIAL","ORDENADO\n\n");
		}else{
			host_print_erro("SEQUENCIAL","FORA DE ORDEM\n\n");
		}
		hipHostFree(h_global_array);
		

	}

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////                        OPEN_MP BUBBLE SORT                             ////////////////////////////

	if(h_global_array_size<=100000 ){
		host_load_input_file(nome);
		double omp_time;	

		omp_time = omp_bubble_sort(h_global_array,h_global_array_size);		
		if(h_is_sort(h_global_array,h_global_array_size)){
			printf("Tempo omp[%3f]\n",omp_time);
			host_print_sucess("OMP_BUBBLE","ORDENADO\n\n");

		}else{
			host_print_erro("OMP_BUBBLE","ORDENADO\n\n");
		}
		hipHostFree(h_global_array);
	}
	
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////                             CUDA HEAP SORT                             ////////////////////////////
	
	if(h_global_nr_threads<=200 && h_global_array_size <=100000){
		host_load_input_file(nome);
		if(d_global_array!= NULL){
			hipFree(d_global_array);
		}
		if(hipMalloc((void**)&d_global_array,h_global_array_size * sizeof(long))){
			host_print_erro("main","Erro ao alocar memória da placa de video");
		}
		printf("Dados copiados para a placa de video %3f MB\n",(double)(h_global_array_size*sizeof(long))/1024/1024);
		hipMemcpy (d_global_array, h_global_array, h_global_array_size*sizeof(long), hipMemcpyHostToDevice);

		double s_time_cuda_heap = wtime();			
			
		KERNEL_set_globals<<<1,1>>>(d_global_array, h_global_array_size,h_global_nr_threads);		
		hipDeviceSynchronize();	
			
		KERNEL_call_sort<<<1,h_global_nr_threads>>>(h_global_nr_threads,CUDA_HEAP);	
		hipDeviceSynchronize();	
		
		host_get_global_array();	
		hipDeviceSynchronize();	
		 
		//Copia as variaveis globais da placa para a memoria do host	
		host_get_global_nr_partitions();
		hipDeviceSynchronize();	

		host_get_global_partitions();
		hipDeviceSynchronize();	 

		
		cpu_merge();
		double e_time_cuda_heap = wtime();	
		if(h_is_sort(h_global_array,h_global_array_size)){
			vet_imprimir(h_global_array,h_global_array_size);		
			printf("Tempo CUDA HEAP[%f]\n",e_time_cuda_heap -s_time_cuda_heap);
			host_print_sucess("CUDA HEAP","ORDENADO\n\n");			
		}
	
		if(d_global_array !=NULL){
			hipFree(d_global_array);
		}
		if(h_global_part!=NULL){		
			hipHostFree(h_global_part);		
		}
		if(h_global_array!=NULL){		
			hipHostFree(h_global_array);		
		}
	
	
	}
	



	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////                           CUDA BUBBLE SORT                             ////////////////////////////
	
	if(h_global_array_size<=100000){

		host_load_input_file(nome);
		if(d_global_array!= NULL){
			hipFree(d_global_array);
		}
		if(hipMalloc((void**)&d_global_array,h_global_array_size * sizeof(long))){
			host_print_erro("main","Erro ao alocar memória da placa de video");
		}
		printf("Dados copiados para a placa de video %3f MB\n",(double)(h_global_array_size*sizeof(long))/1024/1024);
		hipMemcpy (d_global_array, h_global_array, h_global_array_size*sizeof(long), hipMemcpyHostToDevice);
		
		//Cada CUDA core ordena uma partição de d_global_array
		//resulta em um único vetor de partições ordenadas
		double s_time_cuda = wtime();			
		
		KERNEL_set_globals<<<1,1>>>(d_global_array, h_global_array_size,h_global_nr_threads);		
		hipDeviceSynchronize();	
			
		KERNEL_call_sort<<<1,h_global_nr_threads>>>(h_global_nr_threads,CUDA_BUBBLE);	
		hipDeviceSynchronize();		
		double e_time_cuda = wtime();	
		
		
		host_get_global_array();	
		hipDeviceSynchronize();	
		if(h_is_sort(h_global_array,h_global_array_size)){
			vet_imprimir(h_global_array,h_global_array_size);		
			printf("Tempo CUDA BUBBLE[%f]\n",e_time_cuda-s_time_cuda);
			host_print_sucess("CUDA BUBBLE","ORDENADO\n\n");			
		}
		
		double e_time = wtime();			
		if(d_global_array !=NULL){
			hipFree(d_global_array);
		}
		if(h_global_part!=NULL){		
			hipHostFree(h_global_part);		
		}
		if(h_global_array!=NULL){		
			hipHostFree(h_global_array);		
		}
	}

	
	
	return 0;
}



























__host__ void host_intercala (long p, long q, long r, long *v) 
{
   long *w;     
   //printf("p:%ld,r:%ld\nalocando r-p:%ld\n",p,r,r-p);                            //  1
   w =(long *)calloc(r-p,sizeof(long));  //  2
   if(w==NULL){
		host_print_erro("host_intercala","Não foi possivel alocar memoria para w");
   }
   long i = p, j = q;                       //  3
   long k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   
   //vet_imprimir(v,r-p);
   free (w);                               // 12
}

__host__ void host_get_global_array(){
	
	hipHostFree(h_global_array);
	hipHostMalloc((void **)&h_global_array,h_global_array_size*sizeof(long));	
	if(h_global_array==NULL){		
		host_print_erro("host_get_global_array","h_global_array é NULL");
	}	
	KERNEL_get_global_array<<<1,1>>>(h_global_array);	
	hipDeviceSynchronize();	
	//printf("h_global_array[0] %ld\n ",h_global_array[0]);
	
}

__host__ void host_get_global_nr_partitions(){
	long *d_nr_part;
	hipMalloc((void**)&d_nr_part,sizeof(long));	
	KERNEL_get_nr_partitions<<<1,1>>>(d_nr_part);
	hipDeviceSynchronize();
	hipMemcpy(&h_global_nr_part,d_nr_part,sizeof(long),hipMemcpyDeviceToHost);
	//printf("h_global_nr_part %ld\n",h_global_nr_part);
}

__host__ void host_get_global_partitions(){
	
	if(h_global_part!=NULL){
		hipHostFree(h_global_part);
	}
	printf("h_global_nr_part:[%ld]",h_global_nr_threads);
	Data *temp;
	hipHostMalloc((void **)&temp,h_global_nr_threads * sizeof(Data));	
	//hipHostMalloc((void**)&h_global_part,h_global_nr_part* sizeof(Data));
	if(temp==NULL){
		host_print_erro("host_get_global_partitions","Erro ao alocar h_global_part");
	}
	KERNEL_get_array_partitions<<<1,1>>>(temp);
	h_global_part = temp;
	hipDeviceSynchronize();
	//copia o vetor de particoes da placa de video para o host
	//hipMemcpy(&h_global_part[0],&d_part[0],h_global_nr_part * sizeof(Data),hipMemcpyDeviceToHost);	
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE ARQUIVOS                                                                                                            //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ int host_make_input_file(char *nome){

	FILE *fp;

	printf("Gerando arquivo random arquivo.\n");
	if((fp = fopen(nome,"w")) == NULL){		
		host_print_erro("host_make_input_file","Erro na abertura do arquivo");
		return 0;
	}
	long val;
	char buffer[10] = "#size";
	fprintf(fp, "%s\n", buffer);
	fprintf(fp, "%ld\n", h_global_array_size);

	for(int i=0; i<h_global_array_size;i++){
		val  = rand() % 1000000;// (0 <= rand <= 1 Bilhao)
		fprintf(fp, "%ld\n",val);// escreve o numero separado por ','
	}
	printf("Arquivo gerado!\n");
	fclose(fp);
	return 1;
}

__host__ int host_load_input_file(char *nome){
	FILE *fp;

	printf("Abrindo arquivo.\n");
	if((fp = fopen(nome,"r")) == NULL){		
		host_print_erro("host_load_input_file","Erro na leitura do arquivo");
		return 0;
	}
	
	//header
	char buffer[10];
	fscanf(fp, "%s",buffer);
	//printf("-- [%s]\n",buffer);

	//size
	long size;
	fscanf(fp, "%ld",&size);
	//printf("size [%s]:%ld\n",buffer,size);

	long value;
	long i = 0;
	//long *aux = (long*)malloc(size * sizeof(long));
	h_global_array_size = size;
	if(h_global_array!=NULL){
		hipHostFree(h_global_array);
	}
	hipHostMalloc((void **) &h_global_array, h_global_array_size*sizeof(long));	
	//h_global_array = (long*)malloc(size * sizeof(long));
	while ( fscanf(fp, "%ld",&value) != EOF ){		
		h_global_array[i] =(long) value;		
		i++;
	}
	fclose(fp);	
	printf("Arquivo carregado para a memoria!\n");
	return 1;
}

__host__ void swap(long *xp, long *yp)
{
    int temp = *xp;
    *xp = *yp;
    *yp = temp;
}

__host__ void sequencial_bubble_sort(long *arr,  long n)
{
   long i, j;
   for (i = 0; i < n-1; i++){ 
  
       // Last i elements are already in place   
       for (j = 0; j < n-i-1; j++) {
           if (arr[j] > arr[j+1]){
				swap(&arr[j], &arr[j+1]);			
		   }
	   }
	}
              
}


__host__ double omp_bubble_sort(long *arr,  long n){
	long i=0, j=0; 
	long first;
	double start,end;
	start=omp_get_wtime();
	int thr;
	thr = h_global_array_size/2;
	if(h_global_array_size*2>16){
		thr = 16;
	}
	omp_set_num_threads(thr);
	for( i = 0; i < n-1; i++ )
	{
		first = i % 2; 
		#pragma omp parallel for default(none),shared(arr,first,n)
		for( j = first; j < n-1; j += 1 )
		{
			if( arr[ j ] > arr[ j+1 ] )
			{				
				swap( &arr[ j ], &arr[ j+1 ] );
			}
		}
	}
	end=omp_get_wtime();
	return end -start;
}


__host__ void cpu_merge(){

	while(h_global_nr_part>1 ){		
		int count=0;				
		for(int part =0;part<h_global_nr_part;part+=2){			
			int idT = omp_get_thread_num();
			//printf("Thread[%d] mesclando %d e %d\n",idT, part,part+1);
			Data aux_1;
			Data aux_2;
			
			aux_1 = h_global_part[part];
			if(h_global_nr_part%2!=0 && part==h_global_nr_part-1){
				//copia;
				h_global_part[count] =aux_1;
			}else{
				//aux_1 = h_global_part[part];
				aux_2 = h_global_part[part+1];												
				host_intercala(aux_1.a,aux_2.a,aux_2.b+1,&h_global_array[0]);
				Data result;
				result.a=aux_1.a;
				result.b=aux_2.b;
				result.n=aux_1.n+aux_2.n;				
				h_global_part[count] = result;
			}
			count++;
		}						
		h_global_nr_part = ceil((double)h_global_nr_part/(double)2);				
	}
	
}