#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>



__device__ void swap(int *xp, int *yp)  
{  
    int temp = *xp;  
    *xp = *yp;  
    *yp = temp;  
}  
  
__device__ void selectionSort(int *arr, int n)  
{  
    int i, j, min_idx;  
  
    // One by one move boundary of unsorted subarray  
    for (i = 0; i < n-1; i++)  
    {
    	arr[i]  = arr[i] * 100;
        // Find the minimum element in unsorted array  
        /*min_idx = i;  
        for (j = i+1; j < n; j++)  
        if (arr[j] < arr[min_idx])  
            min_idx = j;  
  
        // Swap the found minimum element with the first element  
        swap(&arr[min_idx], &arr[i]);  
        */
    }  
} 


///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// função executada na GPU
__global__ void sort (int *vet_d, int size) {
   int i = threadIdx.x;
   int k=0;
   int part = size / 10; //== cada trede ordenara quatro posições do vetor[40]
   /**
		0 < i=0 < 10 .... 10 < i=1 < 20 .... 20 < i=2 < 30 ... 30 < i=3 < 40(*i)
   */
   int sub_vet_desordenado[4];
   //sub_vet_desordenado = (int)malloc(sizeof(int)*part);
   for(k=i;k<(i*part);k++){
		k=k;   	
   		sub_vet_desordenado[k] = vet_d[k]; 
   }   
   selectionSort(&sub_vet_desordenado[0],part);
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// função executada no HOST

__host__ int *criar_vetor_desordenado(int *v,int size);

__host__ void vet_imprimir(int *v,int size);

int main (int argc, char ** argv) {
	int nthreads = 4;
	int nblocos = 1;

	int size = 40;
	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, size*sizeof(int), hipHostMallocDefault);
	vet_imprimir(vet_desordenado,size); 

	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, size*sizeof(int), hipMemcpyHostToDevice);
	sort<<<1,10>>>(dev_vet, size);
	hipMemcpy (vet_ordenado, dev_vet, size, hipMemcpyDeviceToHost);
	vet_imprimir(vet_ordenado,size); 


	return 0;
}

__host__ int *criar_vetor_desordenado(int *v,int size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	

	hipHostMalloc((void **) &v, size*sizeof(int), hipHostMallocDefault);
	
	//inicia valores do vetor desordenado
	for(int i=0;i<size;i++){
		v[i]= rand() % size;// (0 <= rand <= size)
	}
	return v;
}
__host__ void vet_imprimir(int *v,int size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}

	for(int i=1;i<size-1;i++){
		if(!i%10){
			printf("\n");
		}
		printf("%d\t",v[i]);		
		
	}
	printf("\n");


}
