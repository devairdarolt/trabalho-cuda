#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__host__ int *criar_vetor_desordenado(int *v,int size);

__host__ void vet_imprimir(int *v,int size);

int main (int argc, char ** argv) {
	int nthreads = 4;
	int nblocos = 1;

	//vetores
	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;


	vet_desordenado = criar_vetor_desordenado(vet_desordenado,40);//aloca vetor em host
	vet_imprimir(vet_desordenado,40); 


	return 0;
}

__host__ int *criar_vetor_desordenado(int *v,int size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	

	hipHostMalloc((void **) &v, size*sizeof(int), hipHostMallocDefault);
	
	//inicia valores do vetor desordenado
	for(int i=0;i<size;i++){
		v[i]= rand() % size;// (0 <= rand <= size)
	}
	return v;
}
__host__ void vet_imprimir(int *v,int size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}

	for(int i=1, j=0;i<size-1;i++, j++){
		printf("%d\t",v[i]);		
		if(!j%10){
			printf("\n");
		}
	}
	printf("\n");


}
