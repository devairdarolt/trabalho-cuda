#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>





///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// função executada na GPU
__global__ void sort (int *vet_d, int size) {
   int k = threadIdx.x;   
   int part = size / 10; //== cada trede ordenara quatro posições do vetor[40]
   /**
		0 < i=0 < 4 .... 4 < i=1 < 8 .... 8 < i=2 < 12 ... 12 < i=3 < 18
   */
   /*int min_idx=999999;
   for(k=i*part;k< ((i*part) + part);k++){
		
   		
   } */  

   int a = k*part;
   int b = k*part+part;
   int i=0,j=0;
   int min_idx=0,temp;
   for(i=a;i<b;i++){
   		min_idx = i;
   		for(j=i+1;j<b;j++){
   			if(vet_d[j]<vet_d[min_idx]){
   				min_idx = j;
   			}
   		}
   		temp = 0;
   		temp = vet_d[min_idx];
   		vet_d[min_idx] = vet_d[i];
   		vet_d[i] = temp;	
   }
   
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// função executada no HOST

__host__ int *criar_vetor_desordenado(int *v,int size);

__host__ void vet_imprimir(int *v,int size);

int main (int argc, char ** argv) {
	int nthreads = 4;
	int nblocos = 1;

	int size = 40;
	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, size*sizeof(int), hipHostMallocDefault);
	vet_imprimir(vet_desordenado,size); 

	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, size*sizeof(int), hipMemcpyHostToDevice);
	sort<<<1,10>>>(dev_vet, size);
	hipMemcpy (vet_ordenado, dev_vet, size, hipMemcpyDeviceToHost);
	vet_imprimir(vet_ordenado,size); 


	return 0;
}

__host__ int *criar_vetor_desordenado(int *v,int size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	

	hipHostMalloc((void **) &v, size*sizeof(int), hipHostMallocDefault);
	
	//inicia valores do vetor desordenado
	for(int i=0;i<size;i++){
		v[i]= rand() % size;// (0 <= rand <= size)
	}
	return v;
}
__host__ void vet_imprimir(int *v,int size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}

	printf("\n");
	printf("\n");
	for(int i=0;i<size;i++){
		if(i%10==0){
			printf("\n");
		}
		printf("%d\t",v[i]);		
		
	}
	printf("\n");


}
