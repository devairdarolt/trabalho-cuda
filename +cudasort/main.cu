#include "hip/hip_runtime.h"

#include "lib.h"

#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include <omp.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- VARIÁVEIS GLOBAIS DO HOST                                                                                                      //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


long h_global_nr_part;   //Tamanho do array de particoes;
Data *h_global_part; //Array global para guardar os índices de partições préordenadas
long *h_global_vet_device; //Array global para guardar o vetor a ser ordenado
long h_global_size_vet;
long h_global_nr_nucleos;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DO HOST                                                                                                                //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ void h_intercala (long p, long q, long r, long *v);

__host__ void h_print_erro(const char *func,const char *msg);

__host__ void h_print_sucess(const char *func,const char *msg);

__host__ void host_get_global_vet();

__host__ void get_global_nr_part();

__host__ void get_global_part();

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE ARQUIVO                                                                                                             //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ int criar_arquivo(char *nome);

__host__ int read_file(char *nome);





int main (int argc, char ** argv) {
	//INICIALIZA VARIAVEIS GLOBIAS
	h_global_nr_part=0;   //Tamanho do array de particoes;
	h_global_part=NULL; //Array global para guardar os índices de partições préordenadas
	h_global_vet_device=NULL; //Array global para guardar o vetor a ser ordenado
	
	h_global_size_vet=100000;	
	h_global_nr_nucleos=0;
	


	long nthreads = 96;
	h_global_nr_nucleos = nthreads;
	//long nblocos = 1;
	char nome[] = "teste.map";
	

	
	if (argc == 3 ||argc == 4) {
		nthreads = atoi(argv[1]);
		strcpy(nome,argv[2]);
		if(argc == 4){
			h_global_size_vet = atoi(argv[3]);	
			criar_arquivo(nome);
		}
		//nome = argv[2];
		//h_global_size_vet = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <fileName> --- faz a leitura do arquivo de entrada\n");
		printf ("./main <nthreads> <fileName> <size>--- cria um arquivo de leitura com tamanho size.\n");
		return 0;
	} 
	
	read_file(nome);


	printf("Ordenando vetor de %ld elementos long - %f Kbytes\n",h_global_size_vet,((double)h_global_size_vet*sizeof(long))/(double)1024);	
	//h_global_vet_device =criar_vetor_desordenado(h_global_size_vet);//aloca vetor em host	
	vet_imprimir(h_global_vet_device,h_global_size_vet); 	

	long *dev_vet =NULL;
	int erro = hipMalloc((void**)&dev_vet,h_global_size_vet * sizeof(long));// aloca vetor na memória global da placa
	if(erro){
		printf("\033[0;31m Erro ao alocar memória da placa de video...\n \e[m");
	}
	printf("Dados copiados para a placa de video %3f MB\n",(double)(h_global_size_vet*sizeof(long))/1024/1024);
	hipMemcpy (dev_vet, h_global_vet_device, h_global_size_vet*sizeof(long), hipMemcpyHostToDevice);
	
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	double s_time = wtime();			
	
	GPU_set_globals<<<1,1>>>(dev_vet, h_global_size_vet,nthreads);		
	hipDeviceSynchronize();	
	
	GPU_call_sort<<<1,nthreads>>>(nthreads);	
	hipDeviceSynchronize();	
	double g_time = wtime();	
	h_print_sucess("GPU_call_sort","GPU sort finalizado");
	printf("Tempo levado para ordenar as sub particoes na GPU[%f]\n",g_time-s_time);
	
	
	//Copia as variaveis globais da placa para a memoria do host	
	get_global_nr_part();
	get_global_part();
	host_get_global_vet();

	
	
	
	while(h_global_nr_part>1 ){		
		int count=0;
		omp_set_num_threads(1);//Cria uma thread para cada par de particao, o escalonador que se lasque!
		//printf("\n\n");
		#pragma omp parallel for shared(count,h_global_part,h_global_vet_device)		
		for(int part =0;part<h_global_nr_part;part+=2){			
			int idT = omp_get_thread_num();
			//printf("Thread[%d] mesclando %d e %d\n",idT, part,part+1);
			Data aux_1;
			Data aux_2;
			
			aux_1 = h_global_part[part];
			if(h_global_nr_part%2!=0 && part==h_global_nr_part-1){
				h_global_part[count] =aux_1;
				//printf("%d [%ld -- %ld][%ld] -- cpiado\n",idT,aux_1.a,aux_1.b,aux_1.n);
			}else{
				//aux_1 = h_global_part[part];
				aux_2 = h_global_part[part+1];	
							
				//printf("%d [%ld -- %ld][%ld - %ld][%ld] -- intercalado [%ld -- %ld]\n",count,aux_1.a,aux_1.b,aux_2.a,aux_2.b,aux_1.n+aux_2.n,aux_1.a,aux_2.b);
				h_intercala(aux_1.a,aux_2.a,aux_2.b+1,&h_global_vet_device[0]);
				Data result;
				result.a=aux_1.a;
				result.b=aux_2.b;
				result.n=aux_1.n+aux_2.n;				
				h_global_part[count] = result;
			}
			count++;
		}						
		h_global_nr_part = ceil((double)h_global_nr_part/(double)2);				
	}
	
	printf("\n");
	h_is_sort(h_global_vet_device,h_global_size_vet);

	vet_imprimir(h_global_vet_device,h_global_size_vet);
	printf("\n");
	double e_time = wtime();	
	printf("Tempo total de ordenação:[%f]\n",e_time - s_time);
	//hipFree(dev_vet);
	if(h_global_part!=NULL){
		printf("free h_global_part\n");
		//hipHostFree(h_global_part);
		//free(h_global_part);
	}
	if(h_global_vet_device!=NULL){
		printf("free h_global_vet_device\n");
		//hipHostFree(h_global_vet_device);
		//free(h_global_vet_device);
	}

	return 0;
}
__host__ void h_intercala (long p, long q, long r, long *v) 
{
   long *w;     
   //printf("p:%ld,r:%ld\nalocando r-p:%ld\n",p,r,r-p);                            //  1
   w =(long *)calloc(r-p,sizeof(long));  //  2
   if(w==NULL){
		h_print_erro("h_intercala","Não foi possivel alocar memoria para w");
   }
   long i = p, j = q;                       //  3
   long k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   
   //vet_imprimir(v,r-p);
   free (w);                               // 12
}

__host__ void host_get_global_vet(){
	/* if(h_global_vet_device!=NULL){
		hipHostMalloc((void **)&h_global_vet_device,h_global_size_vet*sizeof(long));	
	} */
	if(h_global_vet_device==NULL){		
		h_print_erro("host_get_global_vet","Erro ao alocar d_vet");
	}	
	GPU_get_global_vet<<<1,1>>>(h_global_vet_device);	
	hipDeviceSynchronize();	
	printf("h_global_vet_device[0] %ld\n ",h_global_vet_device[0]);
	
}

__host__ void get_global_nr_part(){
	long *d_nr_part;
	hipMalloc((void**)&d_nr_part,sizeof(long));	
	GPU_get_nr_part<<<1,1>>>(d_nr_part);
	hipDeviceSynchronize();
	hipMemcpy(&h_global_nr_part,d_nr_part,sizeof(long),hipMemcpyDeviceToHost);
	//printf("h_global_nr_part %ld\n",h_global_nr_part);
}

__host__ void get_global_part(){
	if(h_global_part!=NULL){
		hipHostFree(h_global_part);
	}
	hipHostMalloc((void**)&h_global_part,h_global_nr_part* sizeof(Data));
	if(h_global_part==NULL){
		printf("Erro ao alocar h_global_part\n");
	}
	GPU_get_d_part<<<1,1>>>(h_global_part);
	hipDeviceSynchronize();
	//copia o vetor de particoes da placa de video para o host
	//hipMemcpy(&h_global_part[0],&d_part[0],h_global_nr_part * sizeof(Data),hipMemcpyDeviceToHost);	
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- FUNÇÕES DE ARQUIVOS                                                                                                            //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ int criar_arquivo(char *nome){

	FILE *fp;

	printf("Abrindo arquivo.\n");
	if((fp = fopen(nome,"w")) == NULL){		
		h_print_erro("criar_arquivo","Erro na abertura do arquivo");
		return 0;
	}
	long val;
	char buffer[10] = "#size";
	fprintf(fp, "%s\n", buffer);
	fprintf(fp, "%ld\n", h_global_size_vet);

	for(int i=0; i<h_global_size_vet;i++){
		val  = rand() % 1000;// (0 <= rand <= n)
		fprintf(fp, "%ld\n",val);// escreve o numero separado por ','
	}
	printf("Arquivo gerado!\n");
	fclose(fp);
	return 1;
}

__host__ int read_file(char *nome){
	FILE *fp;

	printf("Abrindo arquivo.\n");
	if((fp = fopen(nome,"r")) == NULL){		
		h_print_erro("read_file","Erro na leitura do arquivo");
		return 0;
	}
	
	//header
	char buffer[10];
	fscanf(fp, "%s",buffer);
	printf("-- [%s]\n",buffer);

	//size
	long size;
	fscanf(fp, "%ld",&size);
	printf("size [%s]:%ld\n",buffer,size);

	long value;
	long i = 0;
	//long *aux = (long*)malloc(size * sizeof(long));
	h_global_size_vet = size;
	hipHostMalloc((void **) &h_global_vet_device, h_global_size_vet*sizeof(long));	
	//h_global_vet_device = (long*)malloc(size * sizeof(long));
	while ( fscanf(fp, "%ld",&value) != EOF ){		
		h_global_vet_device[i] =(long) value;		
		i++;
	}
	fclose(fp);	
	return 1;
}

