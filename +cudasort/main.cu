#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>





///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// função executada na GPU
__global__ void GPU_sort (int *vet_d, int vet_size,int nthreads) {
   

   int k = threadIdx.x;   
   printf("Nucleo %d\n",k );
   int part = vet_size / nthreads; //== cada trede ordenara quatro posições do vetor[40]
   
   /**
		0 < i=0 < 4 .... 4 < i=1 < 8 .... 8 < i=2 < 12 ... 12 < i=3 < 18
   */
   int a = k*part;
   int b = k*part+part;
   int i=0,j=0;
   int min_idx=0,temp;
   for(i=a;i<b;i++){
   		min_idx = i;
   		for(j=i+1;j<b;j++){
   			if(vet_d[j]<vet_d[min_idx]){
   				min_idx = j;
   			}
   		}
   		temp = 0;
   		temp = vet_d[min_idx];
   		vet_d[min_idx] = vet_d[i];
   		vet_d[i] = temp;	
   }
   /*
   for(i=a;i<b;i++){

   		printf("v[%i]:%d\n",i,vet_d[i] );
   }
   printf("\n");
   */
   
   
}


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// função executada no HOST

__host__ int *criar_vetor_desordenado(int *v,int vet_size);

__host__ void vet_imprimir(int *v,int vet_size);

int main (int argc, char ** argv) {
	int nthreads = 4;
	int nblocos = 1;
	int vet_size = 20;


	if (argc == 2) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=4 e vet_size=20\n");
	} 

	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,vet_size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, vet_size*sizeof(int), hipHostMallocDefault);
	printf("Vetor desordenado\n");
	vet_imprimir(vet_desordenado,vet_size); 

	
	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,vet_size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, vet_size*sizeof(int), hipMemcpyHostToDevice);
	//Cada CUDA core ordena uma partição de DEV_VET
	GPU_sort<<<1,nthreads>>>(dev_vet, vet_size,nthreads);
	
	hipDeviceSynchronize();
	hipMemcpy (vet_ordenado, dev_vet, vet_size*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Vetor parcialmente ordenado\n");
	vet_imprimir(vet_ordenado,vet_size); 


	return 0;
}

__host__ int *criar_vetor_desordenado(int *v,int vet_size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	

	hipHostMalloc((void **) &v, vet_size*sizeof(int), hipHostMallocDefault);
	
	//inicia valores do vetor desordenado
	for(int i=0;i<vet_size;i++){
		v[i]= rand() % vet_size;// (0 <= rand <= vet_size)
	}
	return v;
}
__host__ void vet_imprimir(int *v,int vet_size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(vet_size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return;		
	}

	printf("\n");
	printf("\n");
	for(int i=0;i<vet_size;i++){		
		printf("%d\n",v[i]);		
		
	}
	printf("\n");


}
