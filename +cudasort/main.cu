#include "hip/hip_runtime.h"

#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include <omp.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// --- VARIÁVEIS GLOBAIS DO HOST                                                                                                      //
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


long h_global_nr_part;   //Tamanho do array de particoes;
Data * h_global_part; //Array global para guardar os índices de partições préordenadas
long * h_global_vet_device; //Array global para guardar o vetor a ser ordenado
long h_global_size_vet;
long h_global_nr_nucleos;

void h_intercala (long p, long q, long r, long *v);

void h_print_erro(const char *func,const char *msg);

void h_print_sucess(const char *func,const char *msg);


void get_global_vet(){
	
	hipHostMalloc((void **)&h_global_vet_device,h_global_size_vet*sizeof(long));
	
	if(h_global_vet_device==NULL){		
		h_print_erro("get_global_vet","Erro ao alocar d_vet");
	}
	
	GPU_get_global_vet<<<1,1>>>(h_global_vet_device);	
	hipDeviceSynchronize();
	

	printf("h_global_vet_device[0] %ld\n ",h_global_vet_device[0]);
	
}

void get_global_nr_part(){
	long *d_nr_part;
	hipMalloc((void**)&d_nr_part,sizeof(long));	
	GPU_get_nr_part<<<1,1>>>(d_nr_part);
	hipDeviceSynchronize();
	hipMemcpy(&h_global_nr_part,d_nr_part,sizeof(long),hipMemcpyDeviceToHost);
	//printf("h_global_nr_part %ld\n",h_global_nr_part);
}
void get_global_part(){
	Data *d_part;
	hipMalloc((void**)&d_part,h_global_nr_part* sizeof(Data));

	if(d_part==NULL){
		printf("Erro ao alocar d_part\n");
	}
	hipHostMalloc((void**)&h_global_part,h_global_nr_part* sizeof(Data));
	if(h_global_part==NULL){
		printf("Erro ao alocar h_global_part\n");
	}

	GPU_get_d_part<<<1,1>>>(d_part);
	hipDeviceSynchronize();
	//copia o vetor de particoes da placa de video para o host
	hipMemcpy(&h_global_part[0],&d_part[0],h_global_nr_part * sizeof(Data),hipMemcpyDeviceToHost);	
}

int main (int argc, char ** argv) {
	//INICIALIZA VARIAVEIS GLOBIAS
	h_global_nr_part=0;   //Tamanho do array de particoes;
	h_global_part=NULL; //Array global para guardar os índices de partições préordenadas
	h_global_vet_device=NULL; //Array global para guardar o vetor a ser ordenado
	
	h_global_size_vet=100000;	
	h_global_nr_nucleos=0;
	


	long nthreads = 96;
	h_global_nr_nucleos = nthreads;
	//long nblocos = 1;
	

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		h_global_size_vet = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <h_global_size_vet>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=%ld e h_global_size_vet=%ld\n",nthreads,h_global_size_vet);
	} 

	 
	printf("Ordenando vetor de %ld elementos long - %3ld Kbytes\n",h_global_size_vet,(h_global_size_vet*8)/1024);
	//vetores do host	
	
	criar_vetor_desordenado(h_global_vet_device,h_global_size_vet);//aloca vetor em host
	
	//printf("Vetor desordenado\n");
	printf("Teste imprimir..\n");
	vet_imprimir(h_global_vet_device,h_global_size_vet); 

	

	long *dev_vet =NULL;
	int erro = hipMalloc((void**)&dev_vet,h_global_size_vet * sizeof(long));// aloca vetor na memória global da placa
	if(erro){
		printf("\033[0;31m Erro ao alocar memória da placa de video...\n \e[m");
	}
	printf("Dados copiados para a placa de video %3f MB\n",(double)(h_global_size_vet*sizeof(long))/1024/1024);
	hipMemcpy (dev_vet, h_global_vet_device, h_global_size_vet*sizeof(long), hipMemcpyHostToDevice);
	
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	double s_time = wtime();	
		
	
	GPU_set_globals<<<1,1>>>(dev_vet, h_global_size_vet,nthreads);		
	hipDeviceSynchronize();
	
	//printf("Teste de copia vetor grande..n:%ld\n",h_global_size_vet);
	//GPU_print<<<1,1>>>();
	//hipDeviceSynchronize();	
	
	GPU_call_sort<<<1,nthreads>>>(nthreads);	
	hipDeviceSynchronize();	
	double g_time = wtime();	
	h_print_sucess("GPU_call_sort","GPU sort finalizado");
	printf("Tempo levado para ordenar as sub particoes na GPU[%f]\n",g_time-s_time);
	

	

	// MODO 2 - realiza o merge utilizando openMP
	
	
	//Copia as variaveis globais da placa para a memoria do host
	
	get_global_nr_part();
	get_global_part();
	get_global_vet();

	GPU_reset<<<1,1>>>();		
	
	//for(int test=0;test<2;test++){
	while(h_global_nr_part>1 ){
		//printf("h_global_nr_part:%ld\n",h_global_nr_part);
		//h_global_nr_part = ceil((double)h_global_nr_part/(double)2);				
		//Cada duas particao gera uma nova
		int count=0;
		for(int part =0;part<h_global_nr_part;part+=2){
			Data aux_1;
			Data aux_2;
			
			aux_1 = h_global_part[part];
			if(h_global_nr_part%2!=0 && part==h_global_nr_part-1){
				h_global_part[count] =aux_1;
				//printf("%d [%ld -- %ld][%ld] -- cpiado\n",count,aux_1.a,aux_1.b,aux_1.n);
			}else{
				//aux_1 = h_global_part[part];
				aux_2 = h_global_part[part+1];	
							
				//printf("%d [%ld -- %ld][%ld - %ld][%ld] -- intercalado [%ld -- %ld]\n",count,aux_1.a,aux_1.b,aux_2.a,aux_2.b,aux_1.n+aux_2.n,aux_1.a,aux_2.b);
				h_intercala(aux_1.a,aux_2.a,aux_2.b+1,&h_global_vet_device[0]);
				Data result;
				result.a=aux_1.a;
				result.b=aux_2.b;
				result.n=aux_1.n+aux_2.n;				
				h_global_part[count] = result;
			}
			count++;
		}					
		h_global_nr_part = ceil((double)h_global_nr_part/(double)2);				
	}
	
	printf("\n");
	h_is_sort(h_global_vet_device,h_global_size_vet);
	vet_imprimir(h_global_vet_device,h_global_size_vet);
	printf("\n");
	double e_time = wtime();	
	printf("Tempo total de ordenação:[%f]\n",e_time - s_time);
	
	free(h_global_part);
	free(h_global_vet_device);

	return 0;
}
void h_intercala (long p, long q, long r, long *v) 
{
   long *w;     
   //printf("p:%ld,r:%ld\nalocando r-p:%ld\n",p,r,r-p);                            //  1
   w =(long *)malloc((r-p) * sizeof(long));  //  2
   if(w==NULL){
		h_print_erro("h_intercala","Não foi possivel alocar memoria para w");
   }
   long i = p, j = q;                       //  3
   long k = 0;                              //  4

   while (i < q && j < r) {                //  5
      if (v[i] <= v[j])  w[k++] = v[i++];  //  6
      else  w[k++] = v[j++];               //  7
   }                                       //  8
   while (i < q)  w[k++] = v[i++];         //  9
   while (j < r)  w[k++] = v[j++];         // 10
   for (i = p; i < r; ++i)  v[i] = w[i-p]; // 11
   
   //vet_imprimir(v,r-p);
   free (w);                               // 12
}


