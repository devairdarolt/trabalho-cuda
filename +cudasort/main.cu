#include "hip/hip_runtime.h"
#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>




int main (int argc, char ** argv) {
	int nthreads = 3;
	int nblocos = 1;
	int vet_size = 12;

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=%d e vet_size=%d\n",nthreads,vet_size);
	} 

	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,vet_size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, vet_size*sizeof(int));
	//printf("Vetor desordenado\n");
	vet_imprimir(vet_desordenado,vet_size); 

	int *d_nr_part,  h_nr_part;
	hipMalloc((void**)&d_nr_part, sizeof(int));// aloca vetor na memória global da placa
	//Data *d_last_part, h_last_part;
	//hipMalloc((void**)&d_last_part, sizeof(Data));// aloca vetor na memória global da placa

	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,vet_size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, vet_size*sizeof(int), hipMemcpyHostToDevice);
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	double s_time = wtime();	
	
	GPU_set_globals<<<1,1>>>(dev_vet, vet_size,nthreads);		
	hipDeviceSynchronize();

	GPU_sort<<<1,nthreads>>>(nthreads);	
	hipDeviceSynchronize();	
		hipMemcpy (vet_ordenado, dev_vet, vet_size*sizeof(int), hipMemcpyDeviceToHost);			
		vet_imprimir(vet_ordenado,vet_size); 
	//GPU_get_nr_partitions<<<1,1>>>(d_nr_part);// Busca o nr de partições resultantes na operação de sort	
	hipDeviceSynchronize();	
	//hipMemcpy (&h_nr_part, d_nr_part, sizeof(int), hipMemcpyDeviceToHost);
	//printf("particoes para mesclar %d\n",h_nr_part);
	
	while(nthreads>1){
		nthreads = ceil((double)nthreads/(double)2);
		GPU_merge<<<1,nthreads>>>(nthreads);	
		hipDeviceSynchronize();
		hipMemcpy (vet_ordenado, dev_vet, vet_size*sizeof(int), hipMemcpyDeviceToHost);	
		vet_imprimir(vet_ordenado,vet_size); 

	}
	
	
	//GPU_get_nr_partitions<<<1,1>>>(d_nr_part);// Busca o nr de partições resultantes na operação de sort
	//hipDeviceSynchronize();
	//hipMemcpy (&h_nr_part, d_nr_part, sizeof(int), hipMemcpyDeviceToHost);




	hipDeviceSynchronize();
	double e_time = wtime();
	printf("Time:%f (s)\n", e_time-s_time);
	hipMemcpy (vet_ordenado, dev_vet, vet_size*sizeof(int), hipMemcpyDeviceToHost);	
	printf("Vetor parcialmente ordenado\n");
	vet_imprimir(vet_ordenado,vet_size); 
	return 0;
}

