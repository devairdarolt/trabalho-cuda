#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>




int main (int argc, char ** argv) {
	int nthreads = 3;
	int nblocos = 1;
	int vet_size = 12;

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=%d e vet_size=%d\n",nthreads,vet_size);
	} 

	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,vet_size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, vet_size*sizeof(int));
	//printf("Vetor desordenado\n");
	vet_imprimir(vet_desordenado,vet_size); 

	
	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,vet_size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, vet_size*sizeof(int), hipMemcpyHostToDevice);
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	
	//Set global propriedades
	GPU_set_global_prop<<<1,1>>>(dev_vet, vet_size,nthreads);
	
	//Ordena sub arrays
	double s_time = wtime();
	GPU_sort<<<1,nthreads>>>(dev_vet, vet_size,nthreads);			
	hipDeviceSynchronize();
	double e_time = wtime();

	
	//Mescla sub arrays
	GPU_merge<<<1,1>>>(dev_vet, vet_size,nthreads);	
	// Agora precisa fazer o merge entre as partições	
	// Para cada par de partição faça o merge até que reste apenas uma partição
	/*for(int i=nthreads/2;i>=1;i=i/2){
		GPU_merge<<<1,i>>>(dev_vet, vet_size,i);	

	}*/
	printf("Time:%f (s)\n", e_time-s_time);
	hipMemcpy (vet_ordenado, dev_vet, vet_size*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Vetor parcialmente ordenado\n");
	vet_imprimir(vet_ordenado,vet_size); 


	return 0;
}

