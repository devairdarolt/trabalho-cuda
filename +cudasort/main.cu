#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

int *criar_vetor_desordenado(int *v,int size);

void vet_imprimir(int *v,int size);

int main (int argc, char ** argv) {
	int nthreads = 4;
	int nblocos = 1;

	//vetores
	
	int *vet_desordenado, *vet_ordenado;

	vet_desordenado = criar_vetor_desordenado(vet_desordenado,40);
	vet_imprimir(vet_desordenado,40); 


	return 0;
}

int *criar_vetor_desordenado(int *v,int size){

	if(v!=NULL){
		printf("O vetor informado ja existe!\n");
		return v;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
	}
	

	hipHostMalloc((void **) &v, size*sizeof(int));
	
	//inicia valores do vetor desordenado
	for(int i=0;i<size;i++){
		v[i]= rand() % size;// (0 <= rand <= size)
	}
	return v;
}
void vet_imprimir(int *v,int size){
	if(v==NULL){
		printf("O vetor informado é NULL!\n");
		return;
	}
	if(size < 0){
		printf("O tamanho do vetor tem que ser maior que 0\n");
		return		
	}

	for(int i=0;i<size;i++, j++){
		printf("%d\t",v[i]);
		
		if(j==10){
			printf("\n");
		}
	}


}
