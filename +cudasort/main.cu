#include "hip/hip_runtime.h"
#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>




int main (int argc, char ** argv) {
	long nthreads = 3;
	//long nblocos = 1;
	long vet_size = 12;

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=%ld e vet_size=%ld\n",nthreads,vet_size);
	} 
	printf("Ordenando %3ld Kbytes\n",(vet_size*4)/1024);
	//vetores do host	
	long *host_vet=NULL, *device_vet=NULL;
	host_vet = criar_vetor_desordenado(host_vet,vet_size);//aloca vetor em host
	hipHostMalloc((void **) &device_vet, vet_size*sizeof(long));
	//printf("Vetor desordenado\n");
	printf("Vetor criado..\n");
	vet_imprimir(host_vet,vet_size); 

	

	long *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,vet_size * sizeof(long));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, host_vet, vet_size*sizeof(long), hipMemcpyHostToDevice);
	/*for(long i=0;i<vet_size;i++){
		hipMemcpy (&dev_vet[i], &host_vet[i], sizeof(long), hipMemcpyHostToDevice);
	}
	*/
	printf("Dados copiados para a placa de video %3f MB\n",(double)(vet_size*sizeof(long))/1024/1024);
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	double s_time = wtime();	
	
	//hipMemcpy (device_vet, dev_vet, vet_size*sizeof(long), hipMemcpyDeviceToHost);		
	
	GPU_set_globals<<<1,1>>>(dev_vet, vet_size,nthreads);		
	hipDeviceSynchronize();
	
	printf("Teste de copia vetor grande..n:%ld\n",vet_size);
	GPU_print<<<1,1>>>();
	hipDeviceSynchronize();

	printf("\n\n\nGPU_sort\n");
	GPU_sort<<<1,nthreads>>>(nthreads);	
	hipDeviceSynchronize();	
	GPU_print<<<1,1>>>();
	hipDeviceSynchronize();

	/*
	while(nthreads>1){
		
		nthreads = ceil((double)nthreads/(double)2);
		hipDeviceSynchronize();
		GPU_merge<<<1,nthreads>>>(nthreads);	
		hipDeviceSynchronize();
		//printf("\n\n\nPos GPU_merge\n");
		//GPU_print<<<1,1>>>();
		//hipDeviceSynchronize();
	}	
	hipDeviceSynchronize();
	*/
	double e_time = wtime();
	printf("Time:%f (s)\n", e_time-s_time);
	
	printf("\noperacao finalizada\n");
	GPU_print<<<1,1>>>();
	hipDeviceSynchronize();
	
	//free(host_vet);
	hipFree(device_vet);
	GPU_reset<<<1,1>>>();
	return 0;
}

