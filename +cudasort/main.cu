#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>




int main (int argc, char ** argv) {
	int nthreads = 2;
	int nblocos = 1;
	int vet_size = 10;

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=4 e vet_size=20\n");
	} 

	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,vet_size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, vet_size*sizeof(int));
	//printf("Vetor desordenado\n");
	//vet_imprimir(vet_desordenado,vet_size); 

	
	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,vet_size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, vet_size*sizeof(int), hipMemcpyHostToDevice);
	//Cada CUDA core ordena uma partição de DEV_VET
	double s_time = wtime();	
	GPU_sort<<<1,nthreads>>>(dev_vet, vet_size,nthreads);	
	hipDeviceSynchronize();
	double e_time = wtime();
	printf("Time:%f (s)\n", e_time-s_time);
	hipMemcpy (vet_ordenado, dev_vet, vet_size*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Vetor parcialmente ordenado\n");
	vet_imprimir(vet_ordenado,vet_size); 


	return 0;
}

