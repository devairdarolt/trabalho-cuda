#include "lib.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>




int main (int argc, char ** argv) {
	int nthreads = 3;
	int nblocos = 1;
	int vet_size = 12;

	
	if (argc == 3) {
		nthreads = atoi(argv[1]);
		vet_size = atoi(argv[2]);
	}else{
		printf ("./main <nthreads> <vet_size>\n");
		printf ("Caso não haja passagem de parâmetros, nthreads=%d e vet_size=%d\n",nthreads,vet_size);
	} 

	//vetores do host	
	int *vet_desordenado=NULL, *vet_ordenado=NULL;
	vet_desordenado = criar_vetor_desordenado(vet_desordenado,vet_size);//aloca vetor em host
	hipHostMalloc((void **) &vet_ordenado, vet_size*sizeof(int));
	//printf("Vetor desordenado\n");
	vet_imprimir(vet_desordenado,vet_size); 

	
	int *dev_vet =NULL;
	hipMalloc((void**)&dev_vet,vet_size * sizeof(int));// aloca vetor na memória global da placa
	hipMemcpy (dev_vet, vet_desordenado, vet_size*sizeof(int), hipMemcpyHostToDevice);
	//Cada CUDA core ordena uma partição de DEV_VET
	//resulta em um único vetor de partições ordenadas
	double s_time = wtime();	
	GPU_sort<<<1,nthreads>>>(dev_vet, vet_size,nthreads);	
	
	// Agora precisa fazer o merge entre as partições
	GPU_merge<<<1,nthreads>>>(dev_vet, vet_size,nthreads);	

	hipDeviceSynchronize();
	double e_time = wtime();
	printf("Time:%f (s)\n", e_time-s_time);
	hipMemcpy (vet_ordenado, dev_vet, vet_size*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("Vetor parcialmente ordenado\n");
	vet_imprimir(vet_ordenado,vet_size); 


	return 0;
}

